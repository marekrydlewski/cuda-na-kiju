#include "hip/hip_runtime.h"
#include "../catch.hpp"

#define TEST_CUDA_CHECK_RETURN
//--------------------------------------------------------------

#include <vector>

#include "../../GPUPatternMining/Common/MiningCommon.h"

#include "../../GPUPatternMining/PlaneSweep/InstanceTablePlaneSweep.h"

#include "../BaseCudaTestHandler.h"

#include <thrust/device_vector.h>
#include "../../GPUPatternMining/Entities/InstanceTable.h"
//--------------------------------------------------------------

using namespace MiningCommon;
//--------------------------------------------------------------


/*
	Test for graph

	A1-B1-C1-B2-A2-C2
*/ 
TEST_CASE_METHOD(BaseCudaTestHandler,"PlaneSweep_instanceTable | Planesweep main")
{
	unsigned int instancesCount = 6;
	float distanceTreshold = 1;

	std::vector<float> x = { 1, 2, 3, 4, 5, 6 };
	std::vector<float> y = { 1, 1, 1, 1, 1, 1 };
	
	thrust::device_vector<FeatureInstance> instances(instancesCount);
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		instances[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		instances[1] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		instances[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		instances[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		instances[4] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xC;
		instances[5] = fi;
	}

	thrust::device_vector<float> dx = x;
	thrust::device_vector<float> dy = y;

	PlaneSweepTableInstanceResultPtr result = std::make_shared<PlaneSweepTableInstanceResult>();

	PlaneSweep::InstanceTable::PlaneSweep(
		dx
		, dy
		, instances
		, instancesCount
		, distanceTreshold
		, result
	);

	hipDeviceSynchronize();

	{
		thrust::host_vector<unsigned int> indicies = result->indices;
		printf("indices\n");

		for (unsigned int val : indicies)
			printf("%u\n", val);
	}

	Entities::InstanceTable expectedAB(2, 0);
	Entities::InstanceTable expectedAC(1, 2);
	Entities::InstanceTable expectedBC(2, 3);

	std::vector<UInt> resultKeys = {
		0x000A000B
		, 0x000A000C
		, 0x000B000C
	};

	thrust::device_vector<UInt> dResultKeys = resultKeys;


	Entities::InstanceTable* dResults;
	Entities::InstanceTable results[3];

	hipMalloc(reinterpret_cast<void**>(&dResults), 3 * sizeof(Entities::InstanceTable));
	
	result->instanceTableMap->getValues(
		thrust::raw_pointer_cast(dResultKeys.data())
		, dResults
		, 3);

	hipMemcpy(results, dResults, 3 * sizeof(Entities::InstanceTable), hipMemcpyDeviceToHost);

	REQUIRE(results[0].count == expectedAB.count);
	REQUIRE(results[0].startIdx == expectedAB.startIdx);

	REQUIRE(results[1].count == expectedAC.count);
	REQUIRE(results[1].startIdx == expectedAC.startIdx);

	REQUIRE(results[2].count == expectedBC.count);
	REQUIRE(results[2].startIdx == expectedBC.startIdx);
}
// ----------------------------------------------------------------------------

TEST_CASE_METHOD(BaseCudaTestHandler,"PlaneSweep_instanceTable | Planesweep main 1 (Far)")
{
	unsigned int instancesCount = 64;
	float distanceTreshold = 64;

	std::vector<float> x(instancesCount);
	std::vector<float> y(instancesCount);
	
	for (int i = 1; i < 63; ++i)
	{
		x[i] = i;
		y[i] = i * 100;
	}

	x[0] = 0;
	y[0] = 1;

	x[63] = 63;
	y[63] = 1;


	std::vector<FeatureInstance> hostInstances(instancesCount);
	
	for (int i = 1; i < 63; ++i)
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0 + i;
		fi.fields.featureId = 0xB;
		hostInstances[i] = fi;
	}

	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hostInstances[0] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xC;
		hostInstances[63] = fi;
	}


	thrust::device_vector<FeatureInstance> instances = hostInstances;
	thrust::device_vector<float> dx = x;
	thrust::device_vector<float> dy = y;


	PlaneSweepTableInstanceResultPtr result = std::make_shared<PlaneSweepTableInstanceResult>();

	PlaneSweep::InstanceTable::PlaneSweep(
		dx
		, dy
		, instances
		, instancesCount
		, distanceTreshold
		, result
	);

	hipDeviceSynchronize();

	Entities::InstanceTable expectedAC(1, 0);
	
	std::vector<UInt> resultKeys = {
		0x000A000C
	};

	thrust::device_vector<UInt> dResultKeys = resultKeys;

	Entities::InstanceTable* dResults;
	Entities::InstanceTable results[1];

	hipMalloc(reinterpret_cast<void**>(&dResults), 1 * sizeof(Entities::InstanceTable));

	result->instanceTableMap->getValues(
		thrust::raw_pointer_cast(dResultKeys.data())
		, dResults
		, 1);

	hipMemcpy(results, dResults, 1 * sizeof(Entities::InstanceTable), hipMemcpyDeviceToHost);

	REQUIRE(results[0].count == expectedAC.count);
	REQUIRE(results[0].startIdx == expectedAC.startIdx);
}
// ----------------------------------------------------------------------------

/*
	Test for graph

	A0-B0-C0-B1-A1-C1
*/ 
TEST_CASE_METHOD(BaseCudaTestHandler,"PlaneSweep_instanceTable | check countNeighbours function")
{
	UInt instancesCount = 6;

	float hX[] = { 1,2,3,4,5,6 };
	float hY[] = { 1,1,1,1,1,1 };


	thrust::device_vector<FeatureInstance> instances(instancesCount);
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		instances[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		instances[1] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		instances[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		instances[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		instances[4] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xC;
		instances[5] = fi;
	}

	float distanceTreshold = 1.1f;
	float distanceTresholdSquared = 1.1f * 1.1f;

	// tranfering data from host memory to device memory
	float* cX;
	float* cY;
	UInt* cResults;

	hipMalloc(reinterpret_cast<void**>(&cX)		, 6 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&cY)		, 6 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&cResults)	, 6 * uintSize);

	hipMemcpy(cX			, hX		, 6 * sizeof(float)				, hipMemcpyHostToDevice);
	hipMemcpy(cY			, hY		, 6 * sizeof(float)				, hipMemcpyHostToDevice);

	dim3 grid;
	int warpCount = 6; // same as instances count
	findSmallest2D(warpCount * 32, 256, grid.x, grid.y);

	FeatureInstance* cInstances = thrust::raw_pointer_cast(instances.data());

	PlaneSweep::InstanceTable::countNeighbours<<< grid, 256>>> (
		cX
		, cY
		, cInstances
		, instancesCount
		, distanceTreshold
		, distanceTresholdSquared
		, warpCount
		, cResults
	);

	hipDeviceSynchronize();

	UInt hExpected[] = { 0, 1, 1, 1, 1, 1 };
	UInt hResults[6];

	hipMemcpy(hResults, cResults, instancesCount * uintSize, hipMemcpyDeviceToHost);
	
	REQUIRE(std::equal(std::begin(hExpected), std::end(hExpected), hResults));

	hipFree(cX);
	hipFree(cY);
	hipFree(cResults);
}
// ----------------------------------------------------------------------------


TEST_CASE_METHOD(BaseCudaTestHandler,"PlaneSweep_instanceTable | check countNeighbours function (far)")
{
	unsigned int instancesCount = 64;
	float distanceTreshold = 64;

	std::vector<float> x(instancesCount);
	std::vector<float> y(instancesCount);

	for (int i = 1; i < 63; ++i)
	{
		x[i] = i;
		y[i] = i * 100;
	}

	x[0] = 0;
	y[0] = 1;

	x[63] = 63;
	y[63] = 1;


	std::vector<FeatureInstance> hostInstances(instancesCount);

	for (int i = 1; i < 63; ++i)
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0 + i;
		fi.fields.featureId = 0xB;
		hostInstances[i] = fi;
	}

	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hostInstances[0] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hostInstances[63] = fi;
	}


	thrust::device_vector<FeatureInstance> instances = hostInstances;
	thrust::device_vector<float> dx = x;
	thrust::device_vector<float> dy = y;

	thrust::device_vector<UInt> result(instancesCount);

	dim3 grid;
	dim3 block(256, 1, 1);
	int warpCount = instancesCount; // same as instances count
	findSmallest2D(warpCount * 32, 256, grid.x, grid.y);

	FeatureInstance* cInstances = thrust::raw_pointer_cast(instances.data());

	float  distanceTresholdSquared = distanceTreshold * distanceTreshold;

	PlaneSweep::InstanceTable::countNeighbours <<< grid, block >>> (
		thrust::raw_pointer_cast(dx.data())
		, thrust::raw_pointer_cast(dy.data())
		, cInstances
		, instancesCount
		, distanceTreshold
		, distanceTresholdSquared
		, warpCount
		, thrust::raw_pointer_cast(result.data())
		);

	hipDeviceSynchronize();
	
	std::vector<UInt> expected(instancesCount);

	for (int i = 0; i < 63; ++i)
		expected[i] = 0;

	expected[63] = 1;

	thrust::host_vector<UInt> hResult = result;

	REQUIRE(std::equal(hResult.begin(), hResult.end(), expected.begin()));
	
	dx.clear();
	dy.clear();
	result.clear();
}
// ----------------------------------------------------------------------------


TEST_CASE_METHOD(BaseCudaTestHandler,"PlaneSweep_instanceTable | check countNeighbours function (one per warp iteration)")
{
	unsigned int instancesCount = 64;
	float distanceTreshold = 64;

	std::vector<float> x(instancesCount);
	std::vector<float> y(instancesCount);

	for (int i = 1; i < 63; ++i)
	{
		x[i] = i;
		y[i] = i * 100;
	}

	x[0] = 0;
	y[0] = 1;

	x[32] = 32;
	y[32] = 1;

	x[63] = 63;
	y[63] = 1;


	std::vector<FeatureInstance> hostInstances(instancesCount);

	for (int i = 1; i < 63; ++i)
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0 + i;
		fi.fields.featureId = 0xB;
		hostInstances[i] = fi;
	}

	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hostInstances[0] = fi;

		fi.fields.instanceId = 0x2;
		fi.fields.featureId = 0xA;
		hostInstances[32] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hostInstances[63] = fi;
	}


	thrust::device_vector<FeatureInstance> instances = hostInstances;
	thrust::device_vector<float> dx = x;
	thrust::device_vector<float> dy = y;

	thrust::device_vector<UInt> result(instancesCount);

	dim3 grid;
	dim3 block(256, 1, 1);
	int warpCount = instancesCount; // same as instances count
	findSmallest2D(warpCount * 32, 256, grid.x, grid.y);

	FeatureInstance* cInstances = thrust::raw_pointer_cast(instances.data());

	float  distanceTresholdSquared = distanceTreshold * distanceTreshold;

	PlaneSweep::InstanceTable::countNeighbours <<< grid, block >>> (
		thrust::raw_pointer_cast(dx.data())
		, thrust::raw_pointer_cast(dy.data())
		, cInstances
		, instancesCount
		, distanceTreshold
		, distanceTresholdSquared
		, warpCount
		, thrust::raw_pointer_cast(result.data())
		);

	hipDeviceSynchronize();

	std::vector<UInt> expected(instancesCount);

	for (int i = 0; i < 63; ++i)
		expected[i] = 0;

	expected[63] = 2;
	expected[32] = 1;

	thrust::host_vector<UInt> hResult = result;

	REQUIRE(std::equal(hResult.begin(), hResult.end(), expected.begin()));

	dx.clear();
	dy.clear();
	result.clear();
}
// ----------------------------------------------------------------------------


/*
Test for graph

A0-B0-C0-B1-A1-C1
*/
TEST_CASE_METHOD(BaseCudaTestHandler,"PlaneSweep_instanceTable | check findNeighbours function")
{
	// Initialiaze test data

	constexpr UInt instancesCount = 6;

	float hX[] = { 1,2,3,4,5,6 };
	float hY[] = { 1,1,1,1,1,1 };

	thrust::device_vector<FeatureInstance> instances(instancesCount);
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		instances[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		instances[1] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		instances[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		instances[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		instances[4] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xC;
		instances[5] = fi;
	}

	float distanceTreshold = 1.1f;
	float distanceTresholdSquared = 1.1f * 1.1f;

	UInt hScannedResults[] = { 0, 0, 1, 2, 3, 4 };
	constexpr UInt totalPairs = 5;

	// Tranfering data from host memory to device memory

	float* cX;
	float* cY;
	UInt* cStartPositions;
	FeatureInstance* cResultA;
	FeatureInstance* cResultB;

	constexpr UInt resultTableSize = totalPairs;

	hipMalloc(reinterpret_cast<void**>(&cX), 6 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&cY), 6 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&cStartPositions), 6 * uintSize);
	hipMalloc(reinterpret_cast<void**>(&cResultA), resultTableSize * sizeof(FeatureInstance));
	hipMalloc(reinterpret_cast<void**>(&cResultB), resultTableSize * sizeof(FeatureInstance));

	hipMemcpy(cX, hX, 6 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cY, hY, 6 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cStartPositions, hScannedResults, 6 * uintSize, hipMemcpyHostToDevice);

	// Setup startup configuration

	dim3 grid;
	int warpCount = 6; 
	findSmallest2D(warpCount * 32, 256, grid.x, grid.y);

	// run tested function

	FeatureInstance* cInstances = thrust::raw_pointer_cast(instances.data()); 

	PlaneSweep::InstanceTable::findNeighbours << < grid, 256 >> > (
		cX
		, cY
		, cInstances
		, instancesCount
		, distanceTreshold
		, distanceTresholdSquared
		, warpCount
		, cStartPositions
		, cResultA
		, cResultB
	);

	hipDeviceSynchronize();

	// Initialize expected output

	FeatureInstance hExpectedA[totalPairs];
	{
		FeatureInstance fi;
		
		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hExpectedA[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		hExpectedA[1] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		hExpectedA[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hExpectedA[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hExpectedA[4] = fi;
	}

	FeatureInstance hExpectedB[totalPairs];
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		hExpectedB[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		hExpectedB[1] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		hExpectedB[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		hExpectedB[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xC;
		hExpectedB[4] = fi;
	}

	// Fetch result from cuda memory

	FeatureInstance hResultA[totalPairs];
	FeatureInstance hResultB[totalPairs];

	hipMemcpy(hResultA, cResultA, resultTableSize * sizeof(FeatureInstance), hipMemcpyDeviceToHost);
	hipMemcpy(hResultB, cResultB, resultTableSize * sizeof(FeatureInstance), hipMemcpyDeviceToHost);

	// Test output

	REQUIRE(std::equal(std::begin(hExpectedA), std::end(hExpectedA), hResultA));
	REQUIRE(std::equal(std::begin(hExpectedB), std::end(hExpectedB), hResultB));

	// Free allocated resources

	hipFree(cX);
	hipFree(cY);
	hipFree(cStartPositions);
	hipFree(cResultA);
	hipFree(cResultB);
}
// ----------------------------------------------------------------------------


TEST_CASE_METHOD(BaseCudaTestHandler,"PlaneSweep_instanceTable | check findNeighbours function (far)")
{
	unsigned int instancesCount = 64;
	float distanceTreshold = 64;

	std::vector<float> x(instancesCount);
	std::vector<float> y(instancesCount);

	for (int i = 1; i < 63; ++i)
	{
		x[i] = i;
		y[i] = i * 100;
	}

	x[0] = 0;
	y[0] = 1;

	x[63] = 63;
	y[63] = 1;


	std::vector<FeatureInstance> hostInstances(instancesCount);

	for (int i = 1; i < 63; ++i)
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0 + i;
		fi.fields.featureId = 0xB;
		hostInstances[i] = fi;
	}

	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hostInstances[0] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hostInstances[63] = fi;
	}


	thrust::device_vector<FeatureInstance> instances = hostInstances;
	thrust::device_vector<float> dx = x;
	thrust::device_vector<float> dy = y;

	thrust::device_vector<UInt> result(instancesCount);

	dim3 grid;
	dim3 block(256, 1, 1);
	int warpCount = instancesCount; // same as instances count
	findSmallest2D(warpCount * 32, 256, grid.x, grid.y);

	FeatureInstance* cInstances = thrust::raw_pointer_cast(instances.data());

	float  distanceTresholdSquared = distanceTreshold * distanceTreshold;

	std::vector<UInt> startPositions(64, 0);
	thrust::device_vector<UInt> dStartPositions = startPositions;

	thrust::device_vector<FeatureInstance> dResultA(1);
	thrust::device_vector<FeatureInstance> dResultB(1);

	PlaneSweep::InstanceTable::findNeighbours <<< grid, block >>> (
		thrust::raw_pointer_cast(dx.data())
		, thrust::raw_pointer_cast(dy.data())
		, cInstances
		, instancesCount
		, distanceTreshold
		, distanceTresholdSquared
		, warpCount
		, thrust::raw_pointer_cast(dStartPositions.data())
		, thrust::raw_pointer_cast(dResultA.data())
		, thrust::raw_pointer_cast(dResultB.data())
		);

	hipDeviceSynchronize();

	// Initialize expected output

	FeatureInstance hExpectedA[1];
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hExpectedA[0] = fi;
	}

	FeatureInstance hExpectedB[1];
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hExpectedB[0] = fi;
	}

	// Fetch result from cuda memory

	thrust::host_vector<FeatureInstance> resultsA = dResultA;
	thrust::host_vector<FeatureInstance> resultsB = dResultB;

	// Test output

	REQUIRE(std::equal(std::begin(hExpectedA), std::end(hExpectedA), resultsA.begin()));
	REQUIRE(std::equal(std::begin(hExpectedB), std::end(hExpectedB), resultsB.begin()));
}
// ----------------------------------------------------------------------------

TEST_CASE_METHOD(BaseCudaTestHandler,"PlaneSweep_instanceTable | check findNeighbours function (one per warp iteration)")
{
	unsigned int instancesCount = 64;
	float distanceTreshold = 64;

	std::vector<float> x(instancesCount);
	std::vector<float> y(instancesCount);

	for (int i = 1; i < 63; ++i)
	{
		x[i] = i;
		y[i] = i * 100;
	}

	x[0] = 0;
	y[0] = 1;

	x[32] = 32;
	y[32] = 1;

	x[63] = 63;
	y[63] = 1;


	std::vector<FeatureInstance> hostInstances(instancesCount);

	for (int i = 1; i < 63; ++i)
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0 + i;
		fi.fields.featureId = 0xB;
		hostInstances[i] = fi;
	}

	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hostInstances[0] = fi;

		fi.fields.instanceId = 0x2;
		fi.fields.featureId = 0xA;
		hostInstances[32] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hostInstances[63] = fi;
	}


	thrust::device_vector<FeatureInstance> instances = hostInstances;
	thrust::device_vector<float> dx = x;
	thrust::device_vector<float> dy = y;

	thrust::device_vector<UInt> result(instancesCount);

	dim3 grid;
	dim3 block(256, 1, 1);
	int warpCount = instancesCount; // same as instances count
	findSmallest2D(warpCount * 32, 256, grid.x, grid.y);

	FeatureInstance* cInstances = thrust::raw_pointer_cast(instances.data());

	float  distanceTresholdSquared = distanceTreshold * distanceTreshold;

	std::vector<UInt> startPositions(64, 0);
	{
		startPositions[63] = 1;
	}
	thrust::device_vector<UInt> dStartPositions = startPositions;


	thrust::device_vector<FeatureInstance> dResultA(3);
	thrust::device_vector<FeatureInstance> dResultB(3);

	PlaneSweep::InstanceTable::findNeighbours << < grid, block >> > (
		thrust::raw_pointer_cast(dx.data())
		, thrust::raw_pointer_cast(dy.data())
		, cInstances
		, instancesCount
		, distanceTreshold
		, distanceTresholdSquared
		, warpCount
		, thrust::raw_pointer_cast(dStartPositions.data())
		, thrust::raw_pointer_cast(dResultA.data())
		, thrust::raw_pointer_cast(dResultB.data())
		);

	hipDeviceSynchronize();

	// Initialize expected output

	FeatureInstance hExpectedA[3];
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hExpectedA[0] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hExpectedA[1] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hExpectedA[2] = fi;
	}

	FeatureInstance hExpectedB[3];
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x2;
		fi.fields.featureId = 0xA;
		hExpectedB[0] = fi;

		fi.fields.instanceId = 0x2;
		fi.fields.featureId = 0xA;
		hExpectedB[1] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hExpectedB[2] = fi;
	}

	// Fetch result from cuda memory

	thrust::host_vector<FeatureInstance> resultsA = dResultA;
	thrust::host_vector<FeatureInstance> resultsB = dResultB;

	// Test output
	
	//for (int i = 0; i < 3; ++i)
	//{
	//	printf("exp [%#08x, %#08x] res [%#08x, %#08x] \n"
	//		, hExpectedA[i].field, hExpectedB[i].field
	//		, resultsA[i].field  , resultsB[i].field
	//	);
	//}
	

	REQUIRE(std::equal(std::begin(hExpectedA), std::end(hExpectedA), resultsA.begin()));
	REQUIRE(std::equal(std::begin(hExpectedB), std::end(hExpectedB), resultsB.begin()));
}
// ----------------------------------------------------------------------------