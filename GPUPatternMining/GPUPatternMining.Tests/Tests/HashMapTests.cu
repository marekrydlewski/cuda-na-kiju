#include "../catch.hpp"

#include <map>

#include "../../GPUPatternMining/HashMap/gpuhashmapper.h"

#include "../BaseCudaTestHandler.h"
//--------------------------------------------------------------

TEST_CASE_METHOD(BaseCudaTestHandler, "InsertTest", "HashMap")
{
	constexpr size_t threeUINTsize = sizeof(unsigned int) * 3;

	GPUUIntKeyProcessor *intKeyProcessor = new GPUUIntKeyProcessor();
	unsigned int hashSize = 4;

	GPUHashMapper<unsigned int, unsigned int, GPUUIntKeyProcessor> mapper(hashSize, intKeyProcessor);
	mapper.setKeyProcessor(intKeyProcessor);

	unsigned int* c_keys;
	unsigned int* c_values;

	hipMalloc((void**)&c_keys, (sizeof(unsigned int) * 3));
	hipMalloc((void**)&c_values, (sizeof(unsigned int) * 3));

	unsigned int h_keys[] = { 1, 2, 3 };
	unsigned int h_values[] = { 10, 100, 1000 };
	
	hipMemcpy(c_keys, h_keys, threeUINTsize, hipMemcpyHostToDevice);
	hipMemcpy(c_values, h_values, threeUINTsize, hipMemcpyHostToDevice);

	mapper.insertKeyValuePairs(c_keys, c_values, 3);

	hipFree(c_keys);
	hipFree(c_values);

	REQUIRE(true);
}

TEST_CASE_METHOD(BaseCudaTestHandler, "Insert and Read test", "HashMap")
{
	constexpr size_t threeUINTsize = sizeof(unsigned int) * 3;

	GPUUIntKeyProcessor *intKeyProcessor = new GPUUIntKeyProcessor();
	unsigned int hashSize = 4;

	GPUHashMapper<unsigned int, unsigned int, GPUUIntKeyProcessor> mapper(hashSize, intKeyProcessor);
	mapper.setKeyProcessor(intKeyProcessor);

	unsigned int* c_keys;
	unsigned int* c_values;

	hipMalloc((void**)&c_keys, (sizeof(unsigned int) * 3));
	hipMalloc((void**)&c_values, (sizeof(unsigned int) * 3));

	unsigned int h_keys[] = { 1, 2, 3 };
	unsigned int h_values[] = { 10, 100, 1000 };

	hipMemcpy(c_keys, h_keys, threeUINTsize, hipMemcpyHostToDevice);
	hipMemcpy(c_values, h_values, threeUINTsize, hipMemcpyHostToDevice);

	mapper.insertKeyValuePairs(c_keys, c_values, 3);

	unsigned int* c_resultValues;

	hipMalloc((void**)&c_resultValues, (sizeof(unsigned int) * 3));

	unsigned int h_resultValues[] = { 0, 0, 0 };

	mapper.getValues(c_keys, c_resultValues, 3);

	hipMemcpy(h_resultValues, c_resultValues, threeUINTsize, hipMemcpyDeviceToHost);
	
	hipDeviceSynchronize();

	REQUIRE(h_resultValues[0] == h_values[0]);
	REQUIRE(h_resultValues[1] == h_values[1]); 
	REQUIRE(h_resultValues[2] == h_values[2]);

	hipFree(c_keys);
	hipFree(c_values);
	hipFree(c_resultValues);

	REQUIRE(true);
}

TEST_CASE_METHOD(BaseCudaTestHandler, "Insert and Read test with HEX key", "HashMap")
{
	constexpr size_t threeUINTsize = sizeof(unsigned int) * 3;

	GPUUIntKeyProcessor *intKeyProcessor = new GPUUIntKeyProcessor();
	unsigned int hashSize = 4;

	GPUHashMapper<unsigned int, unsigned int, GPUUIntKeyProcessor> mapper(hashSize, intKeyProcessor);
	mapper.setKeyProcessor(intKeyProcessor);

	unsigned int* c_keys;
	unsigned int* c_values;

	hipMalloc((void**)&c_keys, (sizeof(unsigned int) * 3));
	hipMalloc((void**)&c_values, (sizeof(unsigned int) * 3));

	unsigned int h_keys[] = { 0xAA, 0xAB, 0xFF };
	unsigned int h_values[] = { 10, 100, 1000 };

	hipMemcpy(c_keys, h_keys, threeUINTsize, hipMemcpyHostToDevice);
	hipMemcpy(c_values, h_values, threeUINTsize, hipMemcpyHostToDevice);

	mapper.insertKeyValuePairs(c_keys, c_values, 3);

	unsigned int* c_resultValues;

	hipMalloc((void**)&c_resultValues, (sizeof(unsigned int) * 3));

	unsigned int h_resultValues[] = { 0, 0, 0 };

	mapper.getValues(c_keys, c_resultValues, 3);

	hipMemcpy(h_resultValues, c_resultValues, threeUINTsize, hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	REQUIRE(h_resultValues[0] == h_values[0]);
	REQUIRE(h_resultValues[1] == h_values[1]);
	REQUIRE(h_resultValues[2] == h_values[2]);

	hipFree(c_keys);
	hipFree(c_values);
	hipFree(c_resultValues);

	REQUIRE(true);
}