#include "hip/hip_runtime.h"
#include "InstanceTablePlaneSweep.h"

#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "../Common/MiningCommon.h"

#include "../Entities/InstanceTable.h"
#include "../Common/CommonOperations.h"

#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/execution_policy.h>

namespace PlaneSweep
{
	namespace InstanceTable
	{
		__global__ void countNeighbours(
			float* xCoords
			, float* yCoords
			, FeatureInstance* instances
			, int count
			, float radius
			, float radiusSquared
			, int warpsCount
			, UInt* resultNeighboursCount)
		{
			// btid
			int blockThreadId = threadIdx.x;
			//gid
			int globalId = computeLinearAddressFrom2D();
			// wid
			int warpId = globalId / 32;
			// bwid
			int blockWarpId = blockThreadId / 32;
			// wtid
			int warpThreadId = threadIdx.x % 32;

			__shared__ volatile bool * flags;
			__shared__ volatile UInt * found;

			if (blockThreadId == 0)
			{
				flags = static_cast<bool*>(malloc((blockDim.x / 32) * sizeof(bool)));
				found = static_cast<UInt*>(malloc(blockDim.x * sizeof(unsigned int)));
			}

			__syncthreads();

			//uint start= wid        * ((inSize-1 ) / warpCount ) + max(0, - warpCount  + wid          + (inSize - 1) % warpCount ) + 1;
			int start = warpId * ((count - 1) / warpsCount) + max(0, -warpsCount + warpId + (count - 1) % warpsCount) + 1;
			//uint stop=(wid         + 1) * ((inSize-1 ) / warpCount ) + max(0, -warpCount  +  (inSize- 1) % warpCount  + wid          + 1);
			int stop = (warpId + 1) * ((count - 1) / warpsCount) + max(0, -warpsCount + warpId + 1 + (count - 1) % warpsCount);

			if (warpId < warpsCount)
			{
				found[blockThreadId] = 0;

				for (UInt i = start; i <= stop; i++)
				{
					float px = xCoords[i];
					float py = yCoords[i];

					flags[blockWarpId] = false;

					for (int j = i - 32; j >= -32; j -= 32)
					{
						int localId = warpThreadId + j;
						if (localId >= 0)
						{
							if (instances[i].fields.featureId != instances[localId].fields.featureId)
							{

								float lx = xCoords[localId];

								if ((px - lx) > radius)
									flags[blockWarpId] = true;
							

								float ly = yCoords[localId];

								if ((MiningCommon::distance(px, py, lx, ly) <= radiusSquared))
									found[blockThreadId] += 1;
							 }
						}

						if (flags[blockWarpId])
						{
							break;
						}
					}
				}

				MiningCommon::intraWarpReduce(found + blockWarpId * 32);

				if (warpThreadId == 0)
				{
					resultNeighboursCount[warpId] = found[blockWarpId * 32 + 31];
				}
			}

			__syncthreads();

			if (blockThreadId == 0)
			{
				free(const_cast<bool*>(flags));
				free(const_cast<UInt*>(found));
			}
		}
		// --------------------------------------------------------------------------------------------------------------------------------------

		__global__ void findNeighbours(
			float* xCoords
			, float* yCoords
			, FeatureInstance* instances
			, int count
			, float radius
			, float radiusSquared
			, int warpsCount
			, UInt *outStarts
			, FeatureInstance* out_a
			, FeatureInstance* out_b)
		{
			// btid
			int blockThreadId = threadIdx.x;
			//gid
			int globalId = computeLinearAddressFrom2D();
			// wid
			int warpId = globalId / 32;
			// bwid
			int blockWarpId = blockThreadId / 32;
			// wtid
			int warpThreadId = threadIdx.x % 32;

			// const UInt underBuffId = blockWarpId * 64 + warpThreadId;
			// const int aboveBuffId = blockWarpId * 64 + 32 + warpThreadId;

			__shared__ volatile UInt* scanBuf;
			__shared__ volatile bool* flags;
			__shared__ volatile bool* found;
			//volatile __shared__ UInt* buffA;
			//volatile __shared__ UInt* buffB;
			__shared__ UInt* warpBuffPos;


			FeatureInstance temp_a;
			FeatureInstance temp_b;
			// UInt localStart = 0;

			if (blockThreadId == 0)
			{
				// check Allocating http://www.drdobbs.com/parallel/a-massively-parallel-stack-for-data-allo/240162018?pgno=1
				// measure dynamic allocating in different warps
				scanBuf = static_cast<UInt*>(malloc(blockDim.x * sizeof(unsigned int)));
				flags = static_cast<bool*>(malloc((blockDim.x / 32) * sizeof(bool)));
				found = static_cast<bool*>(malloc(blockDim.x * sizeof(bool)));
				//buffA = static_cast<UInt*>(malloc(blockDim.x * sizeof(UInt)));
				//buffB = static_cast<UInt*>(malloc(blockDim.x * sizeof(UInt)));

				warpBuffPos = static_cast<UInt*>(malloc(blockDim.x / 32 * sizeof(UInt)));
			}

			__syncthreads();

			if (warpThreadId == 0)
				warpBuffPos[warpThreadId] = 0;

			//uint start= wid        * ((inSize-1 ) / warpCount ) + max(0, - warpCount  + wid          + (inSize - 1) % warpCount ) + 1;
			int start = warpId * ((count - 1) / warpsCount) + max(0, -warpsCount + warpId + (count - 1) % warpsCount) + 1;
			//uint stop=(wid         + 1) * ((inSize-1 ) / warpCount ) + max(0, -warpCount  +  (inSize- 1) % warpCount  + wid          + 1);
			int stop = (warpId + 1) * ((count - 1) / warpsCount) + max(0, -warpsCount + (count - 1) % warpsCount + warpId + 1);

			if (warpId < warpsCount)
			{
				UInt outStart = outStarts[warpId];

				for (UInt actualRootId = start; actualRootId <= stop; ++actualRootId)
				{
					float px = xCoords[actualRootId];
					float py = yCoords[actualRootId];

					flags[blockWarpId] = false;

					for (int j = actualRootId - 32; j >= -32; j -= 32)
					{
						int localId = warpThreadId + j;
						found[blockThreadId] = false;
						scanBuf[blockThreadId] = 0;

						if (localId >= 0)
						{
							if (instances[actualRootId].fields.featureId != instances[localId].fields.featureId)
							{

								float lx = xCoords[localId];

								if ((px - lx) > radius)
								{
									flags[blockWarpId] = true;
								}

								float ly = yCoords[localId];

								if ((MiningCommon::distance(px, py, lx, ly) <= radiusSquared))
								{
									found[blockThreadId] = true;

									if (instances[actualRootId].field < instances[localId].field)
									{
										temp_a = instances[actualRootId];
										temp_b = instances[localId];
									}
									else
									{
										temp_a = instances[localId];
										temp_b = instances[actualRootId];
									}

									scanBuf[blockThreadId] = 1;
								}
							}
						}


						int lasteL = scanBuf[blockWarpId * 32 + 31];

						MiningCommon::intraWarpScan<UInt>(scanBuf + blockWarpId * 32);
						__syncthreads();

						//if (warpBuffPos[blockWarpId] + scanBuf)

						if (found[blockThreadId])
						{
							int pos = scanBuf[blockThreadId] + outStart;
							out_a[pos] = temp_a;
							out_b[pos] = temp_b;
						}

						outStart += scanBuf[blockWarpId * 32 + 31] + lasteL;

						/*
						scanBuf[blockThreadId] = found[blockThreadId];
						intraWarpScan(scanBuf);

						UInt oldLocalStart = localStart;

						if (found[blockThreadId])
						{
						UInt index = blockWarpId * 64 + (localStart + scanBuf[blockThreadId]) % 64;
						buffA[index] = temp_a.field;
						buffB[index] = temp_b.field;

						}

						// (
						//		localstart
						//		+ last value from scanbuff for last thread in warp
						//		+ last value from found for last thread in warp
						//	) mod 64
						localStart = (localStart + scanBuf[blockWarpId * 32 + 31] + found[blockWarpId * 32 + 31]) % 64;

						if (oldLocalStart < 32 && localStart >= 32)
						{
						out_a[outStart + warpThreadId].field = buffA[underBuffId];
						out_b[outStart + warpThreadId].field = buffB[underBuffId];
						outStart += 32;
						}
						else if (localStart < 32)
						{
						out_a[outStart + warpThreadId].field = buffA[aboveBuffId];
						out_b[outStart + warpThreadId].field = buffB[aboveBuffId];
						outStart += 32;
						}
						*/

						if (flags[blockWarpId])
						{
							break;
						}
					}
				}

				/*
				if (localStart < 32 && warpThreadId < localStart)
				{
				out_a[outStart + warpThreadId].field = buffA[underBuffId];
				out_b[outStart + warpThreadId].field = buffB[underBuffId];
				}
				else if (localStart >= 32 && warpThreadId < localStart - 32)
				{
				out_a[outStart + warpThreadId].field = buffA[aboveBuffId];
				out_b[outStart + warpThreadId].field = buffB[aboveBuffId];
				}
				*/
			}

			__syncthreads();

			if (blockThreadId == 0)
			{
				free(const_cast<UInt*>(scanBuf));
				free(const_cast<bool*>(flags));
				free(const_cast<bool*>(found));
				//free(const_cast<UInt*>(buffA));
				//free(const_cast<UInt*>(buffB));
			}
		}
		// --------------------------------------------------------------------------------------------------------------------------------------

		struct XYFeatureInstanceComparator
		{
			__host__ __device__  bool operator()(
				const  thrust::tuple<float, float, const FeatureInstance>& o1
				, const thrust::tuple<float, float, const  FeatureInstance>& o2)
			{
				return o1.get<0>() < o2.get<0>();
			}
		};

		void SortByXAxis(
			thrust::device_vector<float>& xCoords
			, thrust::device_vector<float>& yCoords
			, thrust::device_vector<FeatureInstance>& instances
		)
		{
			typedef thrust::device_ptr<FeatureInstance> FeatureInstanceIterator;
			typedef thrust::device_ptr<float> FloatIterator;
			typedef thrust::zip_iterator<thrust::tuple<FloatIterator, FloatIterator, FeatureInstanceIterator>> SortIter;

			SortIter begin(thrust::make_tuple(xCoords.begin().base(), yCoords.begin().base(), instances.begin().base()));
			SortIter end(thrust::make_tuple(xCoords.end().base(), yCoords.end().base(), instances.end().base()));

			thrust::sort(thrust::device, begin, end, XYFeatureInstanceComparator());
		}
		// --------------------------------------------------------------------------------------------------------------------------------------

		__host__ void PlaneSweep(
			thrust::device_vector<float>& xCoords
			, thrust::device_vector<float>& yCoords
			, thrust::device_vector<FeatureInstance>& instances
			, UInt count
			, float distanceTreshold
			, PlaneSweepTableInstanceResultPtr result)
		{
			UInt warpsCount = count;
			thrust::device_vector<UInt> neighboursCount(count);
			dim3 grid;

			SortByXAxis(
				xCoords
				, yCoords
				, instances
			);

			findSmallest2D(warpsCount * 32, 256, grid.x, grid.y);

			countNeighbours <<< grid, 256 >>> (
				thrust::raw_pointer_cast(xCoords.data())
				, thrust::raw_pointer_cast(yCoords.data())
				, thrust::raw_pointer_cast(instances.data())
				, count
				, distanceTreshold
				, distanceTreshold * distanceTreshold
				, warpsCount
				, thrust::raw_pointer_cast(neighboursCount.data())
				);

			CUDA_CHECK_RETURN(hipDeviceSynchronize());
			
			UInt totalPairsCount = neighboursCount[count - 1];
			thrust::exclusive_scan(neighboursCount.begin(), neighboursCount.end(), neighboursCount.begin());
			totalPairsCount += neighboursCount[count - 1];

			typedef thrust::device_vector<FeatureInstance> InstancesDeviceVector;

			result->pairsA = InstancesDeviceVector(totalPairsCount);
			result->pairsB = InstancesDeviceVector(totalPairsCount);

			findNeighbours <<< grid, 256 >>> (
				thrust::raw_pointer_cast(xCoords.data())
				, thrust::raw_pointer_cast(yCoords.data())
				, thrust::raw_pointer_cast(instances.data())
				, count
				, distanceTreshold
				, distanceTreshold * distanceTreshold
				, warpsCount
				, neighboursCount.data().get()
				, result->pairsA.data().get()
				, result->pairsB.data().get()
				);

			CUDA_CHECK_RETURN(hipDeviceSynchronize());

			MiningCommon::zipSort(
				result->pairsA
				, result->pairsB
			);

			CUDA_CHECK_RETURN(hipDeviceSynchronize());
			
		}
		// --------------------------------------------------------------------------------------------------------------------------------------
	}
}
