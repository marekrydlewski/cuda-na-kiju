#include "../catch.hpp"

#define TEST_CUDA_CHECK_RETURN
//--------------------------------------------------------------

#include <vector>

#include "../../GPUPatternMining/Common/MiningCommon.h"

#include "../../GPUPatternMining/PlaneSweep/PlaneSweepFoxtrot.h"

#include "../BaseCudaTestHandler.h"

#include <thrust/device_vector.h>
//--------------------------------------------------------------

using namespace MiningCommon;
//--------------------------------------------------------------


/*
	Test for graph

	A1-B1-C1-B2-A2-C2
*/  
TEST_CASE_METHOD(BaseCudaTestHandler, "Planesweep main 0", "PlaneSweep")
{
	unsigned int instancesCount = 6;
	float distanceTreshold = 1;

	std::vector<float> x = { 1, 2, 3, 4, 5, 6 };
	std::vector<float> y = { 1, 1, 1, 1, 1, 1 };
	
	thrust::device_vector<FeatureInstance> instances(instancesCount);
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		instances[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		instances[1] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		instances[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		instances[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		instances[4] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xC;
		instances[5] = fi;
	}

	thrust::device_vector<float> dx = x;
	thrust::device_vector<float> dy = y;

	std::shared_ptr<GPUHashMapper<UInt, NeighboursListInfoHolder, GPUKeyProcessor<UInt>>> hashMap;
	thrust::device_vector<FeatureInstance> resultA;
	thrust::device_vector<FeatureInstance> resultB;

	PlaneSweep::Foxtrot::PlaneSweep(
		dx
		, dy
		, instances
		, instancesCount
		, distanceTreshold
		, hashMap
		, resultA
		, resultB
	);

	NeighboursListInfoHolder expectedA0(1, 0);
	NeighboursListInfoHolder expectedA1(2, 1);
	NeighboursListInfoHolder expectedB0(1, 3);
	NeighboursListInfoHolder expectedB1(1, 4);

	std::vector<UInt> resultKeys = { 
		0x000A0000
		, 0x000A0001
		, 0x000B0000
		, 0x000B0001
	};

	thrust::device_vector<UInt> dResultKeys = resultKeys;


	NeighboursListInfoHolder* dResults;
	NeighboursListInfoHolder results[4];

	hipMalloc(reinterpret_cast<void**>(&dResults), 4 * sizeof(NeighboursListInfoHolder));
	
	hashMap->getValues(
		thrust::raw_pointer_cast(dResultKeys.data())
		, dResults
		, 4);

	hipMemcpy(results, dResults, 4 * sizeof(NeighboursListInfoHolder), hipMemcpyDeviceToHost);

	REQUIRE(results[0].count == expectedA0.count);
	REQUIRE(results[0].count == expectedA0.count);

	REQUIRE(results[1].count == expectedA1.count);
	REQUIRE(results[1].count == expectedA1.count);

	REQUIRE(results[2].count == expectedB0.count);
	REQUIRE(results[2].count == expectedB0.count);
	
	REQUIRE(results[3].count == expectedB1.count);
	REQUIRE(results[3].count == expectedB1.count);
}
// ----------------------------------------------------------------------------


/*
	Test for graph

	A0-B0-C0-B1-A1-C1
*/
TEST_CASE_METHOD(BaseCudaTestHandler, "check countNeighbours function", "PlaneSweep")
{
	UInt instancesCount = 6;

	float hX[] = { 1,2,3,4,5,6 };
	float hY[] = { 1,1,1,1,1,1 };


	thrust::device_vector<FeatureInstance> instances(instancesCount);
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		instances[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		instances[1] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		instances[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		instances[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		instances[4] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xC;
		instances[5] = fi;
	}

	float distanceTreshold = 1.1f;
	float distanceTresholdSquared = 1.1f * 1.1f;

	// tranfering data from host memory to device memory
	float* cX;
	float* cY;
	UInt* cResults;

	hipMalloc(reinterpret_cast<void**>(&cX)		, 6 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&cY)		, 6 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&cResults)	, 6 * uintSize);

	hipMemcpy(cX			, hX		, 6 * sizeof(float)				, hipMemcpyHostToDevice);
	hipMemcpy(cY			, hY		, 6 * sizeof(float)				, hipMemcpyHostToDevice);

	dim3 grid;
	int warpCount = 6; // same as instances count
	findSmallest2D(warpCount * 32, 256, grid.x, grid.y);

	FeatureInstance* cInstances = thrust::raw_pointer_cast(instances.data());

	PlaneSweep::Foxtrot::countNeighbours<<< grid, 256>>> (
		cX
		, cY
		, cInstances
		, instancesCount
		, distanceTreshold
		, distanceTresholdSquared
		, warpCount
		, cResults
	);

	hipDeviceSynchronize();

	UInt hExpected[] = { 0, 1, 1, 1, 1, 1 };
	UInt hResults[6];

	hipMemcpy(hResults, cResults, instancesCount * uintSize, hipMemcpyDeviceToHost);
	
	REQUIRE(std::equal(std::begin(hExpected), std::end(hExpected), hResults));

	hipFree(cX);
	hipFree(cY);
	hipFree(cResults);
}
// ----------------------------------------------------------------------------

/*
Test for graph

A0-B0-C0-B1-A1-C1
*/

TEST_CASE_METHOD(BaseCudaTestHandler, "check findNeighbours function", "PlaneSweep")
{
	// Initialiaze test data

	constexpr UInt instancesCount = 6;

	float hX[] = { 1,2,3,4,5,6 };
	float hY[] = { 1,1,1,1,1,1 };

	thrust::device_vector<FeatureInstance> instances(instancesCount);
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		instances[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		instances[1] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		instances[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		instances[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		instances[4] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xC;
		instances[5] = fi;
	}

	float distanceTreshold = 1.1f;
	float distanceTresholdSquared = 1.1f * 1.1f;

	UInt hScannedResults[] = { 0, 0, 1, 2, 3, 4 };
	constexpr UInt totalPairs = 5;

	// Tranfering data from host memory to device memory

	float* cX;
	float* cY;
	UInt* cStartPositions;
	FeatureInstance* cResultA;
	FeatureInstance* cResultB;

	constexpr UInt resultTableSize = totalPairs;

	hipMalloc(reinterpret_cast<void**>(&cX), 6 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&cY), 6 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&cStartPositions), 6 * uintSize);
	hipMalloc(reinterpret_cast<void**>(&cResultA), resultTableSize * sizeof(FeatureInstance));
	hipMalloc(reinterpret_cast<void**>(&cResultB), resultTableSize * sizeof(FeatureInstance));

	hipMemcpy(cX, hX, 6 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cY, hY, 6 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cStartPositions, hScannedResults, 6 * uintSize, hipMemcpyHostToDevice);

	// Setup startup configuration

	dim3 grid;
	int warpCount = 6; 
	findSmallest2D(warpCount * 32, 256, grid.x, grid.y);

	// run tested function

	FeatureInstance* cInstances = thrust::raw_pointer_cast(instances.data()); 

	PlaneSweep::Foxtrot::findNeighbours << < grid, 256 >> > (
		cX
		, cY
		, cInstances
		, instancesCount
		, distanceTreshold
		, distanceTresholdSquared
		, warpCount
		, cStartPositions
		, cResultA
		, cResultB
	);

	hipDeviceSynchronize();

	// Initialize expected output

	FeatureInstance hExpectedA[totalPairs];
	{
		FeatureInstance fi;
		
		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hExpectedA[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		hExpectedA[1] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		hExpectedA[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hExpectedA[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hExpectedA[4] = fi;
	}

	FeatureInstance hExpectedB[totalPairs];
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		hExpectedB[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		hExpectedB[1] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		hExpectedB[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		hExpectedB[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xC;
		hExpectedB[4] = fi;
	}

	// Fetch result from cuda memory

	FeatureInstance hResultA[totalPairs];
	FeatureInstance hResultB[totalPairs];

	hipMemcpy(hResultA, cResultA, resultTableSize * sizeof(FeatureInstance), hipMemcpyDeviceToHost);
	hipMemcpy(hResultB, cResultB, resultTableSize * sizeof(FeatureInstance), hipMemcpyDeviceToHost);

	// Test output

	REQUIRE(std::equal(std::begin(hExpectedA), std::end(hExpectedA), hResultA));
	REQUIRE(std::equal(std::begin(hExpectedB), std::end(hExpectedB), hResultB));

	// Free allocated resources

	hipFree(cX);
	hipFree(cY);
	hipFree(cStartPositions);
	hipFree(cResultA);
	hipFree(cResultB);
}
// ----------------------------------------------------------------------------