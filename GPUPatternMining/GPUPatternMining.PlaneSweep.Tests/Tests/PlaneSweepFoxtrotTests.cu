#include "hip/hip_runtime.h"
#include "../catch.hpp"

#define TEST_CUDA_CHECK_RETURN
//--------------------------------------------------------------

#include <vector>

#include "../../GPUPatternMining/Common/MiningCommon.h"

#include "../../GPUPatternMining/PlaneSweep/PlaneSweepFoxtrot.h"

#include "../BaseCudaTestHandler.h"

#include <thrust/device_vector.h>
//--------------------------------------------------------------

using namespace MiningCommon;
//--------------------------------------------------------------


/*
	Test for graph

	A1-B1-C1-B2-A2-C2
*/ 
TEST_CASE_METHOD(BaseCudaTestHandler, "Planesweep main 0", "PlaneSweep")
{
	unsigned int instancesCount = 6;
	float distanceTreshold = 1;

	std::vector<float> x = { 1, 2, 3, 4, 5, 6 };
	std::vector<float> y = { 1, 1, 1, 1, 1, 1 };
	
	thrust::device_vector<FeatureInstance> instances(instancesCount);
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		instances[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		instances[1] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		instances[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		instances[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		instances[4] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xC;
		instances[5] = fi;
	}

	thrust::device_vector<float> dx = x;
	thrust::device_vector<float> dy = y;

	std::shared_ptr<GPUHashMapper<UInt, NeighboursListInfoHolder, GPUKeyProcessor<UInt>>> hashMap;
	thrust::device_vector<FeatureInstance> resultA;
	thrust::device_vector<FeatureInstance> resultB;

	PlaneSweep::Foxtrot::PlaneSweep(
		dx
		, dy
		, instances
		, instancesCount
		, distanceTreshold
		, hashMap
		, resultA
		, resultB
	);

	NeighboursListInfoHolder expectedA0(1, 0);
	NeighboursListInfoHolder expectedA1(2, 1);
	NeighboursListInfoHolder expectedB0(1, 3);
	NeighboursListInfoHolder expectedB1(1, 4);

	std::vector<UInt> resultKeys = { 
		0x000A0000
		, 0x000A0001
		, 0x000B0000
		, 0x000B0001
	};

	thrust::device_vector<UInt> dResultKeys = resultKeys;


	NeighboursListInfoHolder* dResults;
	NeighboursListInfoHolder results[4];

	hipMalloc(reinterpret_cast<void**>(&dResults), 4 * sizeof(NeighboursListInfoHolder));
	
	hashMap->getValues(
		thrust::raw_pointer_cast(dResultKeys.data())
		, dResults
		, 4);

	hipMemcpy(results, dResults, 4 * sizeof(NeighboursListInfoHolder), hipMemcpyDeviceToHost);

	REQUIRE(results[0].count == expectedA0.count);
	REQUIRE(results[0].count == expectedA0.count);

	REQUIRE(results[1].count == expectedA1.count);
	REQUIRE(results[1].count == expectedA1.count);

	REQUIRE(results[2].count == expectedB0.count);
	REQUIRE(results[2].count == expectedB0.count);
	
	REQUIRE(results[3].count == expectedB1.count);
	REQUIRE(results[3].count == expectedB1.count);
}
// ----------------------------------------------------------------------------


TEST_CASE_METHOD(BaseCudaTestHandler, "Planesweep main 1 (Far)", "PlaneSweep")
{
	unsigned int instancesCount = 64;
	float distanceTreshold = 64;

	std::vector<float> x(instancesCount);
	std::vector<float> y(instancesCount);
	
	for (int i = 1; i < 63; ++i)
	{
		x[i] = i;
		y[i] = i * 100;
	}

	x[0] = 0;
	y[0] = 1;

	x[63] = 63;
	y[63] = 1;


	std::vector<FeatureInstance> hostInstances(instancesCount);
	
	for (int i = 1; i < 63; ++i)
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0 + i;
		fi.fields.featureId = 0xB;
		hostInstances[i] = fi;
	}

	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hostInstances[0] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hostInstances[63] = fi;
	}


	thrust::device_vector<FeatureInstance> instances = hostInstances;
	thrust::device_vector<float> dx = x;
	thrust::device_vector<float> dy = y;

	std::shared_ptr<GPUHashMapper<UInt, NeighboursListInfoHolder, GPUKeyProcessor<UInt>>> hashMap;
	thrust::device_vector<FeatureInstance> resultA;
	thrust::device_vector<FeatureInstance> resultB;

	PlaneSweep::Foxtrot::PlaneSweep(
		dx
		, dy
		, instances
		, instancesCount
		, distanceTreshold
		, hashMap
		, resultA
		, resultB
	);

	NeighboursListInfoHolder expectedA0(1, 0);
	
	std::vector<UInt> resultKeys = {
		0x000A0000
	};

	thrust::device_vector<UInt> dResultKeys = resultKeys;

	NeighboursListInfoHolder* dResults;
	NeighboursListInfoHolder results[1];

	hipMalloc(reinterpret_cast<void**>(&dResults), 1 * sizeof(NeighboursListInfoHolder));

	hashMap->getValues(
		thrust::raw_pointer_cast(dResultKeys.data())
		, dResults
		, 1);

	hipMemcpy(results, dResults, 1 * sizeof(NeighboursListInfoHolder), hipMemcpyDeviceToHost);

	REQUIRE(results[0].count == expectedA0.count);
}
// ----------------------------------------------------------------------------

/*
	Test for graph

	A0-B0-C0-B1-A1-C1
*/ 
TEST_CASE_METHOD(BaseCudaTestHandler, "check countNeighbours function", "PlaneSweep")
{
	UInt instancesCount = 6;

	float hX[] = { 1,2,3,4,5,6 };
	float hY[] = { 1,1,1,1,1,1 };


	thrust::device_vector<FeatureInstance> instances(instancesCount);
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		instances[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		instances[1] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		instances[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		instances[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		instances[4] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xC;
		instances[5] = fi;
	}

	float distanceTreshold = 1.1f;
	float distanceTresholdSquared = 1.1f * 1.1f;

	// tranfering data from host memory to device memory
	float* cX;
	float* cY;
	UInt* cResults;

	hipMalloc(reinterpret_cast<void**>(&cX)		, 6 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&cY)		, 6 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&cResults)	, 6 * uintSize);

	hipMemcpy(cX			, hX		, 6 * sizeof(float)				, hipMemcpyHostToDevice);
	hipMemcpy(cY			, hY		, 6 * sizeof(float)				, hipMemcpyHostToDevice);

	dim3 grid;
	int warpCount = 6; // same as instances count
	findSmallest2D(warpCount * 32, 256, grid.x, grid.y);

	FeatureInstance* cInstances = thrust::raw_pointer_cast(instances.data());

	PlaneSweep::Foxtrot::countNeighbours<<< grid, 256>>> (
		cX
		, cY
		, cInstances
		, instancesCount
		, distanceTreshold
		, distanceTresholdSquared
		, warpCount
		, cResults
	);

	hipDeviceSynchronize();

	UInt hExpected[] = { 0, 1, 1, 1, 1, 1 };
	UInt hResults[6];

	hipMemcpy(hResults, cResults, instancesCount * uintSize, hipMemcpyDeviceToHost);
	
	REQUIRE(std::equal(std::begin(hExpected), std::end(hExpected), hResults));

	hipFree(cX);
	hipFree(cY);
	hipFree(cResults);
}
// ----------------------------------------------------------------------------


TEST_CASE_METHOD(BaseCudaTestHandler, "check countNeighbours function (far)", "PlaneSweep")
{
	unsigned int instancesCount = 64;
	float distanceTreshold = 64;

	std::vector<float> x(instancesCount);
	std::vector<float> y(instancesCount);

	for (int i = 1; i < 63; ++i)
	{
		x[i] = i;
		y[i] = i * 100;
	}

	x[0] = 0;
	y[0] = 1;

	x[63] = 63;
	y[63] = 1;


	std::vector<FeatureInstance> hostInstances(instancesCount);

	for (int i = 1; i < 63; ++i)
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0 + i;
		fi.fields.featureId = 0xB;
		hostInstances[i] = fi;
	}

	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hostInstances[0] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hostInstances[63] = fi;
	}


	thrust::device_vector<FeatureInstance> instances = hostInstances;
	thrust::device_vector<float> dx = x;
	thrust::device_vector<float> dy = y;

	thrust::device_vector<UInt> result(instancesCount);

	dim3 grid;
	dim3 block(256, 1, 1);
	int warpCount = instancesCount; // same as instances count
	findSmallest2D(warpCount * 32, 256, grid.x, grid.y);

	FeatureInstance* cInstances = thrust::raw_pointer_cast(instances.data());

	float  distanceTresholdSquared = distanceTreshold * distanceTreshold;

	PlaneSweep::Foxtrot::countNeighbours <<< grid, block >>> (
		thrust::raw_pointer_cast(dx.data())
		, thrust::raw_pointer_cast(dy.data())
		, cInstances
		, instancesCount
		, distanceTreshold
		, distanceTresholdSquared
		, warpCount
		, thrust::raw_pointer_cast(result.data())
		);

	hipDeviceSynchronize();
	
	std::vector<UInt> expected(instancesCount);

	for (int i = 0; i < 63; ++i)
		expected[i] = 0;

	expected[63] = 1;

	thrust::host_vector<UInt> hResult = result;

	REQUIRE(std::equal(hResult.begin(), hResult.end(), expected.begin()));
	
	dx.clear();
	dy.clear();
	result.clear();
}
// ----------------------------------------------------------------------------


TEST_CASE_METHOD(BaseCudaTestHandler, "check countNeighbours function (one per warp iteration)", "PlaneSweep")
{
	unsigned int instancesCount = 64;
	float distanceTreshold = 64;

	std::vector<float> x(instancesCount);
	std::vector<float> y(instancesCount);

	for (int i = 1; i < 63; ++i)
	{
		x[i] = i;
		y[i] = i * 100;
	}

	x[0] = 0;
	y[0] = 1;

	x[32] = 32;
	y[32] = 1;

	x[63] = 63;
	y[63] = 1;


	std::vector<FeatureInstance> hostInstances(instancesCount);

	for (int i = 1; i < 63; ++i)
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0 + i;
		fi.fields.featureId = 0xB;
		hostInstances[i] = fi;
	}

	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hostInstances[0] = fi;

		fi.fields.instanceId = 0x2;
		fi.fields.featureId = 0xA;
		hostInstances[32] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hostInstances[63] = fi;
	}


	thrust::device_vector<FeatureInstance> instances = hostInstances;
	thrust::device_vector<float> dx = x;
	thrust::device_vector<float> dy = y;

	thrust::device_vector<UInt> result(instancesCount);

	dim3 grid;
	dim3 block(256, 1, 1);
	int warpCount = instancesCount; // same as instances count
	findSmallest2D(warpCount * 32, 256, grid.x, grid.y);

	FeatureInstance* cInstances = thrust::raw_pointer_cast(instances.data());

	float  distanceTresholdSquared = distanceTreshold * distanceTreshold;

	PlaneSweep::Foxtrot::countNeighbours <<< grid, block >>> (
		thrust::raw_pointer_cast(dx.data())
		, thrust::raw_pointer_cast(dy.data())
		, cInstances
		, instancesCount
		, distanceTreshold
		, distanceTresholdSquared
		, warpCount
		, thrust::raw_pointer_cast(result.data())
		);

	hipDeviceSynchronize();

	std::vector<UInt> expected(instancesCount);

	for (int i = 0; i < 63; ++i)
		expected[i] = 0;

	expected[63] = 2;
	expected[32] = 1;

	thrust::host_vector<UInt> hResult = result;

	REQUIRE(std::equal(hResult.begin(), hResult.end(), expected.begin()));

	dx.clear();
	dy.clear();
	result.clear();
}
// ----------------------------------------------------------------------------


/*
Test for graph

A0-B0-C0-B1-A1-C1
*/
TEST_CASE_METHOD(BaseCudaTestHandler, "check findNeighbours function", "PlaneSweep")
{
	// Initialiaze test data

	constexpr UInt instancesCount = 6;

	float hX[] = { 1,2,3,4,5,6 };
	float hY[] = { 1,1,1,1,1,1 };

	thrust::device_vector<FeatureInstance> instances(instancesCount);
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		instances[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		instances[1] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		instances[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		instances[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		instances[4] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xC;
		instances[5] = fi;
	}

	float distanceTreshold = 1.1f;
	float distanceTresholdSquared = 1.1f * 1.1f;

	UInt hScannedResults[] = { 0, 0, 1, 2, 3, 4 };
	constexpr UInt totalPairs = 5;

	// Tranfering data from host memory to device memory

	float* cX;
	float* cY;
	UInt* cStartPositions;
	FeatureInstance* cResultA;
	FeatureInstance* cResultB;

	constexpr UInt resultTableSize = totalPairs;

	hipMalloc(reinterpret_cast<void**>(&cX), 6 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&cY), 6 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&cStartPositions), 6 * uintSize);
	hipMalloc(reinterpret_cast<void**>(&cResultA), resultTableSize * sizeof(FeatureInstance));
	hipMalloc(reinterpret_cast<void**>(&cResultB), resultTableSize * sizeof(FeatureInstance));

	hipMemcpy(cX, hX, 6 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cY, hY, 6 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cStartPositions, hScannedResults, 6 * uintSize, hipMemcpyHostToDevice);

	// Setup startup configuration

	dim3 grid;
	int warpCount = 6; 
	findSmallest2D(warpCount * 32, 256, grid.x, grid.y);

	// run tested function

	FeatureInstance* cInstances = thrust::raw_pointer_cast(instances.data()); 

	PlaneSweep::Foxtrot::findNeighbours << < grid, 256 >> > (
		cX
		, cY
		, cInstances
		, instancesCount
		, distanceTreshold
		, distanceTresholdSquared
		, warpCount
		, cStartPositions
		, cResultA
		, cResultB
	);

	hipDeviceSynchronize();

	// Initialize expected output

	FeatureInstance hExpectedA[totalPairs];
	{
		FeatureInstance fi;
		
		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hExpectedA[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		hExpectedA[1] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		hExpectedA[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hExpectedA[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hExpectedA[4] = fi;
	}

	FeatureInstance hExpectedB[totalPairs];
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		hExpectedB[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		hExpectedB[1] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		hExpectedB[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		hExpectedB[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xC;
		hExpectedB[4] = fi;
	}

	// Fetch result from cuda memory

	FeatureInstance hResultA[totalPairs];
	FeatureInstance hResultB[totalPairs];

	hipMemcpy(hResultA, cResultA, resultTableSize * sizeof(FeatureInstance), hipMemcpyDeviceToHost);
	hipMemcpy(hResultB, cResultB, resultTableSize * sizeof(FeatureInstance), hipMemcpyDeviceToHost);

	// Test output

	REQUIRE(std::equal(std::begin(hExpectedA), std::end(hExpectedA), hResultA));
	REQUIRE(std::equal(std::begin(hExpectedB), std::end(hExpectedB), hResultB));

	// Free allocated resources

	hipFree(cX);
	hipFree(cY);
	hipFree(cStartPositions);
	hipFree(cResultA);
	hipFree(cResultB);
}
// ----------------------------------------------------------------------------


TEST_CASE_METHOD(BaseCudaTestHandler, "check findNeighbours function (far)", "PlaneSweep")
{
	unsigned int instancesCount = 64;
	float distanceTreshold = 64;

	std::vector<float> x(instancesCount);
	std::vector<float> y(instancesCount);

	for (int i = 1; i < 63; ++i)
	{
		x[i] = i;
		y[i] = i * 100;
	}

	x[0] = 0;
	y[0] = 1;

	x[63] = 63;
	y[63] = 1;


	std::vector<FeatureInstance> hostInstances(instancesCount);

	for (int i = 1; i < 63; ++i)
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0 + i;
		fi.fields.featureId = 0xB;
		hostInstances[i] = fi;
	}

	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hostInstances[0] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hostInstances[63] = fi;
	}


	thrust::device_vector<FeatureInstance> instances = hostInstances;
	thrust::device_vector<float> dx = x;
	thrust::device_vector<float> dy = y;

	thrust::device_vector<UInt> result(instancesCount);

	dim3 grid;
	dim3 block(256, 1, 1);
	int warpCount = instancesCount; // same as instances count
	findSmallest2D(warpCount * 32, 256, grid.x, grid.y);

	FeatureInstance* cInstances = thrust::raw_pointer_cast(instances.data());

	float  distanceTresholdSquared = distanceTreshold * distanceTreshold;

	std::vector<UInt> startPositions(64, 0);
	thrust::device_vector<UInt> dStartPositions = startPositions;

	thrust::device_vector<FeatureInstance> dResultA(1);
	thrust::device_vector<FeatureInstance> dResultB(1);

	PlaneSweep::Foxtrot::findNeighbours <<< grid, block >>> (
		thrust::raw_pointer_cast(dx.data())
		, thrust::raw_pointer_cast(dy.data())
		, cInstances
		, instancesCount
		, distanceTreshold
		, distanceTresholdSquared
		, warpCount
		, thrust::raw_pointer_cast(dStartPositions.data())
		, thrust::raw_pointer_cast(dResultA.data())
		, thrust::raw_pointer_cast(dResultB.data())
		);

	hipDeviceSynchronize();

	// Initialize expected output

	FeatureInstance hExpectedA[1];
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hExpectedA[0] = fi;
	}

	FeatureInstance hExpectedB[1];
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hExpectedB[0] = fi;
	}

	// Fetch result from cuda memory

	thrust::host_vector<FeatureInstance> resultsA = dResultA;
	thrust::host_vector<FeatureInstance> resultsB = dResultB;

	// Test output

	REQUIRE(std::equal(std::begin(hExpectedA), std::end(hExpectedA), resultsA.begin()));
	REQUIRE(std::equal(std::begin(hExpectedB), std::end(hExpectedB), resultsB.begin()));
}
// ----------------------------------------------------------------------------


TEST_CASE_METHOD(BaseCudaTestHandler, "check findNeighbours function (one per warp iteration)", "PlaneSweep")
{
	unsigned int instancesCount = 64;
	float distanceTreshold = 64;

	std::vector<float> x(instancesCount);
	std::vector<float> y(instancesCount);

	for (int i = 1; i < 63; ++i)
	{
		x[i] = i;
		y[i] = i * 100;
	}

	x[0] = 0;
	y[0] = 1;

	x[32] = 32;
	y[32] = 1;

	x[63] = 63;
	y[63] = 1;


	std::vector<FeatureInstance> hostInstances(instancesCount);

	for (int i = 1; i < 63; ++i)
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0 + i;
		fi.fields.featureId = 0xB;
		hostInstances[i] = fi;
	}

	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hostInstances[0] = fi;

		fi.fields.instanceId = 0x2;
		fi.fields.featureId = 0xA;
		hostInstances[32] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hostInstances[63] = fi;
	}


	thrust::device_vector<FeatureInstance> instances = hostInstances;
	thrust::device_vector<float> dx = x;
	thrust::device_vector<float> dy = y;

	thrust::device_vector<UInt> result(instancesCount);

	dim3 grid;
	dim3 block(256, 1, 1);
	int warpCount = instancesCount; // same as instances count
	findSmallest2D(warpCount * 32, 256, grid.x, grid.y);

	FeatureInstance* cInstances = thrust::raw_pointer_cast(instances.data());

	float  distanceTresholdSquared = distanceTreshold * distanceTreshold;

	std::vector<UInt> startPositions(64, 0);
	{
		startPositions[63] = 1;
	}
	thrust::device_vector<UInt> dStartPositions = startPositions;


	thrust::device_vector<FeatureInstance> dResultA(3);
	thrust::device_vector<FeatureInstance> dResultB(3);

	PlaneSweep::Foxtrot::findNeighbours << < grid, block >> > (
		thrust::raw_pointer_cast(dx.data())
		, thrust::raw_pointer_cast(dy.data())
		, cInstances
		, instancesCount
		, distanceTreshold
		, distanceTresholdSquared
		, warpCount
		, thrust::raw_pointer_cast(dStartPositions.data())
		, thrust::raw_pointer_cast(dResultA.data())
		, thrust::raw_pointer_cast(dResultB.data())
		);

	hipDeviceSynchronize();

	// Initialize expected output

	FeatureInstance hExpectedA[3];
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hExpectedA[0] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hExpectedA[1] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hExpectedA[2] = fi;
	}

	FeatureInstance hExpectedB[3];
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x2;
		fi.fields.featureId = 0xA;
		hExpectedB[0] = fi;

		fi.fields.instanceId = 0x2;
		fi.fields.featureId = 0xA;
		hExpectedB[1] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hExpectedB[2] = fi;
	}

	// Fetch result from cuda memory

	thrust::host_vector<FeatureInstance> resultsA = dResultA;
	thrust::host_vector<FeatureInstance> resultsB = dResultB;

	// Test output
	
	//for (int i = 0; i < 3; ++i)
	//{
	//	printf("exp [%#08x, %#08x] res [%#08x, %#08x] \n"
	//		, hExpectedA[i].field, hExpectedB[i].field
	//		, resultsA[i].field  , resultsB[i].field
	//	);
	//}
	

	REQUIRE(std::equal(std::begin(hExpectedA), std::end(hExpectedA), resultsA.begin()));
	REQUIRE(std::equal(std::begin(hExpectedB), std::end(hExpectedB), resultsB.begin()));
}
// ----------------------------------------------------------------------------