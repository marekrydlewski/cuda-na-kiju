#include "hip/hip_runtime.h"
#include "AnyLengthInstancesUniquePrevalenceProvider.h"
#include <thrust/execution_policy.h>
#include <thrust/unique.h>
#include <algorithm>


AnyLengthInstancesUniquePrevalenceProvider::AnyLengthInstancesUniquePrevalenceProvider(
	TypesCountsMapResultPtr typesCountsMap)
	: typesCountsMap(typesCountsMap)
{

}

std::shared_ptr<thrust::device_vector<float>> AnyLengthInstancesUniquePrevalenceProvider::getPrevalenceFromCandidatesInstances(
	Entities::GpuCliques cliquesCandidates
	, InstanceTree::InstanceTreeResultPtr instanceTreeResult
) const
{
	const unsigned int candidatesCount = cliquesCandidates.candidatesCount;
	const unsigned int instancesCount = instanceTreeResult->instancesCliqueId.size();

	auto result = std::make_shared<thrust::device_vector<float>>(candidatesCount, 0.f);

	if (candidatesCount == 0 || instancesCount == 0)
		return result;
	
	thrust::device_vector<unsigned int> cliquesID(candidatesCount);
	thrust::device_vector<unsigned int> instancesCounts(candidatesCount);

	// how many candidates "make it" as instances
	unsigned int existingCandidatesCount = thrust::reduce_by_key(
		thrust::device
		, instanceTreeResult->instancesCliqueId.begin()
		, instanceTreeResult->instancesCliqueId.end()
		, thrust::constant_iterator<unsigned int>(1)
		, cliquesID.begin()
		, instancesCounts.begin()
	).first - cliquesID.begin();

	thrust::device_vector<unsigned int> begins(existingCandidatesCount);

	thrust::exclusive_scan(
		thrust::device
		, instancesCounts.begin()
		, instancesCounts.begin() + existingCandidatesCount
		, begins.begin()
	);

	thrust::device_vector<unsigned int> idxs(existingCandidatesCount);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	thrust::sequence(
		thrust::device
		, idxs.begin()
		, idxs.end()
	);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	thrust::device_vector<FeatureInstance> levelTempStorage(instancesCount);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	auto cliquesTypesCount = getTypesCountOnGpuForCliquesCandidates(
		cliquesCandidates, typesCountsMap->map
	);

	MinimalCandidatePrevalenceCounter prevalenceCounter;
	{
		prevalenceCounter.data = instanceTreeResult->instances.data();
		prevalenceCounter.begins = begins.data();
		prevalenceCounter.typeCount = cliquesTypesCount->data();
		prevalenceCounter.counts = instancesCounts.data();
		prevalenceCounter.cliqueIds = cliquesID.data();
		prevalenceCounter.levelUniquesTempStorage = levelTempStorage.data();

		prevalenceCounter.results = result->data();
		prevalenceCounter.levelsCount = cliquesCandidates.currentCliquesSize;
		prevalenceCounter.instancesCount = instancesCount;
		prevalenceCounter.candidatesCount = cliquesCandidates.candidatesCount;
	}

	unsigned int countPerIteration = 10;
	unsigned int currentStart = 0;
	unsigned int currentEnd = existingCandidatesCount % countPerIteration;

	currentEnd = std::min(currentEnd + countPerIteration, existingCandidatesCount);

	while (currentEnd <= existingCandidatesCount)
	{
		thrust::for_each(
			thrust::device
			, idxs.begin() + currentStart
			, idxs.begin() + currentEnd
			, prevalenceCounter);
		currentStart += countPerIteration;
		currentEnd += countPerIteration;

		CUDA_CHECK_RETURN(hipDeviceSynchronize());
	}
	
	return result;
}
