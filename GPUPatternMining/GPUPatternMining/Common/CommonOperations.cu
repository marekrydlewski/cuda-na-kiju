#include "hip/hip_runtime.h"
#include "CommonOperations.h"


namespace MiningCommon
{
	__global__ void InsertIntoHashMap(
		GPUFeatureInstanceHashMapBean bean,
		FeatureInstance* keys,
		unsigned int* deltas,
		unsigned int* counts,
		unsigned int count
	)
	{
		unsigned int tid = computeLinearAddressFrom2D();

		if (tid < count)
		{
			GPUHashMapperProcedures::insertKeyValuePair(
				bean,
				keys[tid].field,
				NeighboursListInfoHolder(counts[tid], deltas[tid])
			);
		}
	}
	//---------------------------------------------------------------------------------------------

	void zipSort(thrust::device_vector<FeatureInstance>& a, thrust::device_vector<FeatureInstance>& b)
	{
		typedef thrust::device_ptr<FeatureInstance> FeatureInstanceIterator;
		typedef thrust::tuple<FeatureInstanceIterator, FeatureInstanceIterator> Tcc;
		typedef thrust::zip_iterator<Tcc> OutputZipIterator;

		FeatureInstanceIterator aBegin = a.begin().base();
		FeatureInstanceIterator bBegin = b.begin().base();

		FeatureInstanceIterator aEnd = a.end().base();
		FeatureInstanceIterator bEnd = a.end().base();

		OutputZipIterator begin(thrust::make_tuple(aBegin, bBegin));
		OutputZipIterator end(thrust::make_tuple(aEnd, bEnd));

		thrust::sort(begin, end, FeatureInstanceComparator());
	}
	//---------------------------------------------------------------------------------------------

}