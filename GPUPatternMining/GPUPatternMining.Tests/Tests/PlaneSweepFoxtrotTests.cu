#include "../catch.hpp"

#define TEST_CUDA_CHECK_RETURN
//--------------------------------------------------------------

#include "../../GPUPatternMining/PlaneSweep/PlaneSweepFoxtrot.cu"

#include "../../GPUPatternMining/MiningCommon.h"

#include "../BaseCudaTestHandler.h"

#include <thrust/device_vector.h>
//--------------------------------------------------------------

/*
	Test for graph

	A1-B1-C1-B2-A2-C2
*/
/*
TEST_CASE_METHOD(BaseCudaTestHandler, "check first neighbours list element (neigbours count)", "PlaneSweep")
{
	float x[] = { 1,2,3,4,5,6 };
	float y[] = { 1,2,3,4,5,6 };
	unsigned int type[] = { 0xA, 0xB, 0xC, 0xB, 0xA, 0xC };
	unsigned int ids[] = {1, 1, 1, 2, 2, 2 };
	unsigned int instancesCount = 6;
	unsigned int distanceTreshold = 1;

	GPUUIntKeyProcessor *intKeyProcessor = new GPUUIntKeyProcessor();

	UIntTableGpuHashMap hashMap(6, intKeyProcessor);

	PlaneSweep::Foxtrot::PlaneSweep<float>(x, y, type, ids, instancesCount, distanceTreshold, hashMap);

	unsigned int h_resultKeys[] = { 0x000A000B, 0x000A000C, 0x000B000C };
	unsigned int* c_resultKey;

	hipMalloc(reinterpret_cast<void**>(&c_resultKey), sizeof(3 * uintSize));
	hipMemcpy(c_resultKey, h_resultKeys, 3 * uintSize, hipMemcpyHostToDevice);

	unsigned int** c_results;
	unsigned int* d_results[3]; // pointers to GPU memory in host memory
	unsigned int h_result;

	hipMalloc(reinterpret_cast<void**>(&c_results), 3 * uintPtrSize);
	hashMap.getValues(c_resultKey, c_results, 3);

	hipMemcpy(d_results, c_results, 3 * uintPtrSize, hipMemcpyDeviceToHost);

	hipMemcpy(&h_result, d_results[0], uintSize, hipMemcpyDeviceToHost);
	REQUIRE(h_result == 2); // |A-B|

	hipMemcpy(&h_result, d_results[1], uintSize, hipMemcpyDeviceToHost);
	REQUIRE(h_result == 1); // |A-C|

	hipMemcpy(&h_result, d_results[3], uintSize, hipMemcpyDeviceToHost);
	REQUIRE(h_result == 2); // |B-C|
}
// ----------------------------------------------------------------------------
*/


/*
	Test for graph

	A0-B0-C0-B1-A1-C1
*/
TEST_CASE_METHOD(BaseCudaTestHandler, "check countNeighbours function", "PlaneSweep")
{
	UInt instancesCount = 6;

	float hX[] = { 1,2,3,4,5,6 };
	float hY[] = { 1,1,1,1,1,1 };


	thrust::device_vector<FeatureInstance> instances(instancesCount);
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		instances[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		instances[1] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		instances[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		instances[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		instances[4] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xC;
		instances[5] = fi;
	}

	float distanceTreshold = 1.1;
	float distanceTresholdSquared = 1.1 * 1.1;

	// tranfering data from host memory to device memory
	float* cX;
	float* cY;
	UInt* cResults;

	hipMalloc(reinterpret_cast<void**>(&cX)		, 6 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&cY)		, 6 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&cResults)	, 6 * uintSize);

	hipMemcpy(cX			, hX		, 6 * sizeof(float)				, hipMemcpyHostToDevice);
	hipMemcpy(cY			, hY		, 6 * sizeof(float)				, hipMemcpyHostToDevice);

	dim3 grid;
	int warpCount = 6; // same as instances count
	findSmallest2D(warpCount * 32, 256, grid.x, grid.y);

	FeatureInstance* cInstances = thrust::raw_pointer_cast(instances.data());

	PlaneSweep::Foxtrot::countNeighbours<<< grid, 256>>> (
		cX
		, cY
		, cInstances
		, instancesCount
		, distanceTreshold
		, distanceTresholdSquared
		, warpCount
		, cResults
	);

	hipDeviceSynchronize();

	UInt hExpected[] = { 0, 1, 1, 1, 1, 1 };
	UInt hResults[6];

	hipMemcpy(hResults, cResults, instancesCount * uintSize, hipMemcpyDeviceToHost);
	
	REQUIRE(std::equal(std::begin(hExpected), std::end(hExpected), hResults));

	hipFree(cX);
	hipFree(cY);
	hipFree(cResults);
}
// ----------------------------------------------------------------------------


/*
Test for graph

A0-B0-C0-B1-A1-C1
*/

TEST_CASE_METHOD(BaseCudaTestHandler, "check findNeighbours function", "PlaneSweep")
{
	// Initialiaze test data

	constexpr UInt instancesCount = 6;

	float hX[] = { 1,2,3,4,5,6 };
	float hY[] = { 1,1,1,1,1,1 };

	thrust::device_vector<FeatureInstance> instances(instancesCount);
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		instances[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		instances[1] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		instances[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		instances[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		instances[4] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xC;
		instances[5] = fi;
	}

	float distanceTreshold = 1.1;
	float distanceTresholdSquared = 1.1 * 1.1;

	UInt hScannedResults[] = { 0, 0, 1, 2, 3, 4 };
	constexpr UInt totalPairs = 5;

	// Tranfering data from host memory to device memory

	float* cX;
	float* cY;
	UInt* cStartPositions;
	FeatureInstance* cResultA;
	FeatureInstance* cResultB;

	constexpr UInt resultTableSize = totalPairs;

	hipMalloc(reinterpret_cast<void**>(&cX), 6 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&cY), 6 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&cStartPositions), 6 * uintSize);
	hipMalloc(reinterpret_cast<void**>(&cResultA), resultTableSize * sizeof(FeatureInstance));
	hipMalloc(reinterpret_cast<void**>(&cResultB), resultTableSize * sizeof(FeatureInstance));

	hipMemcpy(cX, hX, 6 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cY, hY, 6 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cStartPositions, hScannedResults, 6 * uintSize, hipMemcpyHostToDevice);

	// Setup startup configuration

	dim3 grid;
	int warpCount = 6; 
	findSmallest2D(warpCount * 32, 256, grid.x, grid.y);

	// run tested function

	FeatureInstance* cInstances = thrust::raw_pointer_cast(instances.data()); 

	PlaneSweep::Foxtrot::findNeighbours << < grid, 256 >> > (
		cX
		, cY
		, cInstances
		, instancesCount
		, distanceTreshold
		, distanceTresholdSquared
		, warpCount
		, cStartPositions
		, cResultA
		, cResultB
	);

	hipDeviceSynchronize();

	// Initialize expected output

	FeatureInstance hExpectedA[totalPairs];
	{
		FeatureInstance fi;
		
		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xA;
		hExpectedA[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		hExpectedA[1] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		hExpectedA[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hExpectedA[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xA;
		hExpectedA[4] = fi;
	}

	FeatureInstance hExpectedB[totalPairs];
	{
		FeatureInstance fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xB;
		hExpectedB[0] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		hExpectedB[1] = fi;

		fi.fields.instanceId = 0x0;
		fi.fields.featureId = 0xC;
		hExpectedB[2] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xB;
		hExpectedB[3] = fi;

		fi.fields.instanceId = 0x1;
		fi.fields.featureId = 0xC;
		hExpectedB[4] = fi;
	}

	// Fetch result from cuda memory

	FeatureInstance hResultA[totalPairs];
	FeatureInstance hResultB[totalPairs];

	hipMemcpy(hResultA, cResultA, resultTableSize * sizeof(FeatureInstance), hipMemcpyDeviceToHost);
	hipMemcpy(hResultB, cResultB, resultTableSize * sizeof(FeatureInstance), hipMemcpyDeviceToHost);

	// Test output

	REQUIRE(std::equal(std::begin(hExpectedA), std::end(hExpectedA), hResultA));
	REQUIRE(std::equal(std::begin(hExpectedB), std::end(hExpectedB), hResultB));

	// Free allocated resources

	hipFree(cX);
	hipFree(cY);
	hipFree(cStartPositions);
	hipFree(cResultA);
	hipFree(cResultB);
}
// ----------------------------------------------------------------------------