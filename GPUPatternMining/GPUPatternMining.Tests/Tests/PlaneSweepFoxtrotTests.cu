#include "../catch.hpp"

#define TEST_CUDA_CHECK_RETURN
//--------------------------------------------------------------

#include "../../GPUPatternMining/PlaneSweep/PlaneSweepFoxtrot.cu"

#include "../../GPUPatternMining/MiningCommon.h"

#include "../BaseCudaTestHandler.h"
//--------------------------------------------------------------

/*
	Test for graph

	A1-B1-C1-B2-A2-C2
*/
/*
TEST_CASE_METHOD(BaseCudaTestHandler, "check first neighbours list element (neigbours count)", "PlaneSweep")
{
	float x[] = { 1,2,3,4,5,6 };
	float y[] = { 1,2,3,4,5,6 };
	unsigned int type[] = { 0xA, 0xB, 0xC, 0xB, 0xA, 0xC };
	unsigned int ids[] = {1, 1, 1, 2, 2, 2 };
	unsigned int instancesCount = 6;
	unsigned int distanceTreshold = 1;

	GPUUIntKeyProcessor *intKeyProcessor = new GPUUIntKeyProcessor();

	UIntTableGpuHashMap hashMap(6, intKeyProcessor);

	PlaneSweep::Foxtrot::PlaneSweep<float>(x, y, type, ids, instancesCount, distanceTreshold, hashMap);

	unsigned int h_resultKeys[] = { 0x000A000B, 0x000A000C, 0x000B000C };
	unsigned int* c_resultKey;

	hipMalloc(reinterpret_cast<void**>(&c_resultKey), sizeof(3 * uintSize));
	hipMemcpy(c_resultKey, h_resultKeys, 3 * uintSize, hipMemcpyHostToDevice);

	unsigned int** c_results;
	unsigned int* d_results[3]; // pointers to GPU memory in host memory
	unsigned int h_result;

	hipMalloc(reinterpret_cast<void**>(&c_results), 3 * uintPtrSize);
	hashMap.getValues(c_resultKey, c_results, 3);

	hipMemcpy(d_results, c_results, 3 * uintPtrSize, hipMemcpyDeviceToHost);

	hipMemcpy(&h_result, d_results[0], uintSize, hipMemcpyDeviceToHost);
	REQUIRE(h_result == 2); // |A-B|

	hipMemcpy(&h_result, d_results[1], uintSize, hipMemcpyDeviceToHost);
	REQUIRE(h_result == 1); // |A-C|

	hipMemcpy(&h_result, d_results[3], uintSize, hipMemcpyDeviceToHost);
	REQUIRE(h_result == 2); // |B-C|
}
// ----------------------------------------------------------------------------
*/

TEST_CASE_METHOD(BaseCudaTestHandler, "check countNeighbours function", "PlaneSweep")
{
	float hX[] = { 1,2,3,4,5,6 };
	float hY[] = { 1,1,1,1,1,1 };
	UInt hTypes[] = { 0xA, 0xB, 0xC, 0xB, 0xA, 0xC };
	UInt hIds[] = { 1, 1, 1, 2, 2, 2 };
	UInt instancesCount = 6;
	float distanceTreshold = 1.1;
	float distanceTresholdSquared = 1.1 * 1.1;

	// tranfering data from host memory to device memory
	float* cX;
	float* cY;
	UInt* cType;
	UInt* cIds;
	UInt* cResults;

	hipMalloc(reinterpret_cast<void**>(&cX)		, 6 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&cY)		, 6 * sizeof(float));
	hipMalloc(reinterpret_cast<void**>(&cType)		, 6 * uintSize);
	hipMalloc(reinterpret_cast<void**>(&cIds)		, 6 * uintSize);
	hipMalloc(reinterpret_cast<void**>(&cResults)	, 6 * uintSize);

	hipMemcpy(cX	, hX	, 6 * sizeof(float)	, hipMemcpyHostToDevice);
	hipMemcpy(cY	, hY	, 6 * sizeof(float)	, hipMemcpyHostToDevice);
	hipMemcpy(cType, hTypes, 6 * uintSize		, hipMemcpyHostToDevice);
	hipMemcpy(cIds	, hIds	, 6 * uintSize		, hipMemcpyHostToDevice);


	dim3 grid;
	int wc = 6;
	findSmallest2D(wc * 32, 256, grid.x, grid.y);

	PlaneSweep::Foxtrot::countNeighbours<<< grid, 256>>> (cX, cY, cType, cIds, instancesCount, distanceTreshold, distanceTresholdSquared, cResults, wc);

	UInt hExpected[] = { 1, 2, 2, 2, 2, 1 };
	UInt hResults[6];

	hipMemcpy(hResults, cResults, instancesCount, hipMemcpyDeviceToHost);

	for (int i = 0; i < 6; ++i)
		printf("%d = %u\n", i, hResults[i]);
	
	REQUIRE(std::equal(std::begin(hExpected), std::end(hExpected), hResults));

	hipFree(cX);
	hipFree(cY);
	hipFree(cType);
	hipFree(cIds);
	hipFree(cResults);
}
// ----------------------------------------------------------------------------