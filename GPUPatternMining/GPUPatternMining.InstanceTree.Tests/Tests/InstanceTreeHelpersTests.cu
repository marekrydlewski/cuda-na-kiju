#include "..\catch.hpp"
#include "..\BaseCudaTestHandler.h"

#define TEST_CUDA_CHECK_RETURN
//---------------------------------------------------------------------------------------------------

#include "..\..\GPUPatternMining/InstanceTree/InstanceTreeHelpers.h"
#include "../../GPUPatternMining/InstanceTree/IntanceTablesMapCreator.h"
//---------------------------------------------------------------------------------------------------

using namespace InstanceTreeHelpers;
//---------------------------------------------------------------------------------------------------

typedef thrust::device_vector<unsigned int> UIntThrustVector;
typedef std::shared_ptr<UIntThrustVector> UIntThrustVectorPtr;

typedef thrust::device_vector<FeatureInstance> FeatureInstanceThrustVector;
typedef std::shared_ptr<FeatureInstanceThrustVector> FeatureInstanceThrustVectorPtr;
//---------------------------------------------------------------------------------------------------

TEST_CASE_METHOD(BaseCudaTestHandler, "Instance tree helpers | insert first pair count")
{
	thrust::device_vector<unsigned int> keys;
	{
		std::vector<unsigned int> hKeys
		{
			0x000A000B
			, 0x000A000C
			, 0x000B000C
		};

		keys = hKeys;
	}

	thrust::device_vector<Entities::InstanceTable> values;
	{
		std::vector<Entities::InstanceTable> hValues;

		Entities::InstanceTable it;

		it.count = 2;
		it.startIdx = 0;
		hValues.push_back(it);

		it.count = 3;
		it.startIdx = 2;
		hValues.push_back(it);

		it.count = 6;
		it.startIdx = 5;
		hValues.push_back(it);

		values = hValues;
	}
	
	auto proc = GPUUIntKeyProcessor();
	auto map = IntanceTablesMapCreator::InstanceTableMap(5, &proc);


	map.insertKeyValuePairs(
		thrust::raw_pointer_cast(keys.data())
		, thrust::raw_pointer_cast(values.data())
		, 3
	);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	thrust::device_vector<unsigned int> result(3);

	thrust::device_vector<thrust::device_vector<unsigned short>> cliquesData;
	thrust::host_vector<thrust::device_vector<unsigned short>> hcliquesData;
	{
		
		std::vector<unsigned short> first = { 0x000A, 0x000B, 0x000C };
		std::vector<unsigned short> second = { 0x000A, 0x000C, 0x000D };
		std::vector<unsigned short> third = { 0x000B, 0x000C, 0x000D };

		hcliquesData.push_back(first);
		hcliquesData.push_back(second);
		hcliquesData.push_back(third);

		cliquesData = hcliquesData;
	}

	thrust::device_vector<thrust::device_ptr<const unsigned short>> cliques;
	{
		std::vector<thrust::device_ptr<const unsigned short>> hcliques;
		for (const thrust::device_vector<unsigned short>& vec : hcliquesData)
			hcliques.push_back(vec.data());

		cliques = hcliques;
	}

	dim3 insertGrid;
	findSmallest2D(3, 256, insertGrid.x, insertGrid.y);
	
	fillFirstPairCountFromMap <<< insertGrid, 256 >>>(
		map.getBean()
		, thrust::raw_pointer_cast(cliques.data())
		, 3
		, result.data()
	);

	hipDeviceSynchronize();

	thrust::host_vector<unsigned int> hResult = result;

	std::vector<unsigned int> expected =
	{
		2, 3, 6
	};

	REQUIRE(std::equal(expected.begin(), expected.end(), hResult.begin()));
}
//--------------------------------------------------------------


TEST_CASE_METHOD(BaseCudaTestHandler, "Instance tree helpers | for groups simple")
{

	thrust::device_vector<unsigned int> counts;
	{
		std::vector<unsigned int> hCount = {
			2, 3, 2, 1
		};

		counts = hCount;
	}

	auto result = forGroups(counts);

	std::vector<unsigned int> expectedGroupNumbers = {
		0, 0, 1, 1, 1, 2, 2, 3
	};

	std::vector<unsigned int> expectedItemNumbers = {
		0, 1, 0, 1, 2, 0, 1, 0
	};

	thrust::host_vector<unsigned int> resultGroupNumbers = result->groupNumbers;
	thrust::host_vector<unsigned int> resultItemNumbers = result->itemNumbers;

	REQUIRE(std::equal(expectedGroupNumbers.begin(), expectedGroupNumbers.end(), resultGroupNumbers.begin()));
	REQUIRE(std::equal(expectedItemNumbers.begin(), expectedItemNumbers.end(), resultItemNumbers.begin()));
	REQUIRE(result->threadCount == 8);
}
//--------------------------------------------------------------

TEST_CASE_METHOD(BaseCudaTestHandler, "Instance tree helpers | for groups last zero")
{

	thrust::device_vector<unsigned int> counts;
	{
		std::vector<unsigned int> hCount = {
			2, 3, 2, 0
		};

		counts = hCount;
	}

	auto result = forGroups(counts);

	std::vector<unsigned int> expectedGroupNumbers = {
		0, 0, 1, 1, 1, 2, 2
	};

	std::vector<unsigned int> expectedItemNumbers = {
		0, 1, 0, 1, 2, 0, 1
	};

	thrust::host_vector<unsigned int> resultGroupNumbers = result->groupNumbers;
	thrust::host_vector<unsigned int> resultItemNumbers = result->itemNumbers;

	REQUIRE(std::equal(expectedGroupNumbers.begin(), expectedGroupNumbers.end(), resultGroupNumbers.begin()));
	REQUIRE(std::equal(expectedItemNumbers.begin(), expectedItemNumbers.end(), resultItemNumbers.begin()));
	REQUIRE(result->threadCount == 7);
}
//--------------------------------------------------------------

TEST_CASE_METHOD(BaseCudaTestHandler, "Instance tree helpers | for groups inner zeros")
{

	thrust::device_vector<unsigned int> counts;
	{
		std::vector<unsigned int> hCount = {
			1, 2, 0, 0, 1
		};

		counts = hCount;
	}

	auto result = forGroups(counts);

	std::vector<unsigned int> expectedGroupNumbers = {
		0, 1, 1, 4
	};

	std::vector<unsigned int> expectedItemNumbers = {
		0, 0, 1, 0
	};

	thrust::host_vector<unsigned int> resultGroupNumbers = result->groupNumbers;
	thrust::host_vector<unsigned int> resultItemNumbers = result->itemNumbers;

	REQUIRE(std::equal(expectedGroupNumbers.begin(), expectedGroupNumbers.end(), resultGroupNumbers.begin()));
	REQUIRE(std::equal(expectedItemNumbers.begin(), expectedItemNumbers.end(), resultItemNumbers.begin()));
	REQUIRE(result->threadCount == 4);
}
//--------------------------------------------------------------

TEST_CASE_METHOD(BaseCudaTestHandler, "Instance tree helpers | for groups inner zeros extended")
{

	thrust::device_vector<unsigned int> counts;
	{
		std::vector<unsigned int> hCount = {
			1, 2, 0, 0, 4, 1
		};

		counts = hCount;
	}

	auto result = forGroups(counts);

	std::vector<unsigned int> expectedGroupNumbers = {
		0, 1, 1, 4, 4, 4, 4, 5
	};

	std::vector<unsigned int> expectedItemNumbers = {
		0, 0, 1, 0, 1, 2, 3, 0
	};

	thrust::host_vector<unsigned int> resultGroupNumbers = result->groupNumbers;
	thrust::host_vector<unsigned int> resultItemNumbers = result->itemNumbers;

	REQUIRE(std::equal(expectedGroupNumbers.begin(), expectedGroupNumbers.end(), resultGroupNumbers.begin()));
	REQUIRE(std::equal(expectedItemNumbers.begin(), expectedItemNumbers.end(), resultItemNumbers.begin()));
	REQUIRE(result->threadCount == expectedGroupNumbers.size());
}
//--------------------------------------------------------------

/*
Test for graph

         C3
          |
A1-B1-C1-B2-A2-C2
*/
TEST_CASE_METHOD(BaseCudaTestHandler, "Instance tree helpers | insert first two levels")
{
	thrust::device_vector<FeatureInstance> pairsA;
	thrust::device_vector<FeatureInstance> pairsB;
	{
		
		//a1 - b1
		//a2 - b2
		//a2 - c2
		//b1 - c1
		//b2 - c1
		//b2 - c3
		
		std::vector<FeatureInstance> hPairsA = {
			{ 0x000A0001 }
			,{ 0x000A0002 }
			,{ 0x000A0002 }
			,{ 0x000B0001 }
			,{ 0x000B0002 }
			,{ 0x000B0002 }
		};

		std::vector<FeatureInstance> hPairsB = {
			{ 0x000B0001 }
			,{ 0x000B0002 }
			,{ 0x000C0002 }
			,{ 0x000C0001 }
			,{ 0x000C0001 }
			,{ 0x000C0003 }
		};

		pairsA = hPairsA;
		pairsB = hPairsB;
	}

	auto instanceMapResult = IntanceTablesMapCreator::createTypedNeighboursListMap(
		pairsA
		, pairsB
	);

	thrust::device_vector<thrust::device_vector<unsigned short>> cliquesData;
	thrust::host_vector<thrust::device_vector<unsigned short>> hcliquesData;
	{

		std::vector<unsigned short> first = { 0x000A, 0x000B, 0x000C };
		std::vector<unsigned short> second = { 0x000B, 0x000C, 0x000D };

		hcliquesData.push_back(first);
		hcliquesData.push_back(second);

		cliquesData = hcliquesData;
	}

	thrust::device_vector<thrust::device_ptr<const unsigned short>> cliques;
	{
		std::vector<thrust::device_ptr<const unsigned short>> hcliques;
		for (const thrust::device_vector<unsigned short>& vec : hcliquesData)
			hcliques.push_back(vec.data());

		cliques = hcliques;
	}

	thrust::device_vector<unsigned int> groupNumber;
	{
		std::vector<unsigned int> hGroupNumber
		{
			0, 0, 1, 1, 1
		};

		groupNumber = hGroupNumber;
	}

	thrust::device_vector<unsigned int> itemNumber;
	{
		std::vector<unsigned int> hItemNumber
		{
			0, 1, 0, 1, 2
		};

		itemNumber = hItemNumber;
	}

	dim3 insertGrid;
	findSmallest2D(5, 256, insertGrid.x, insertGrid.y);


	thrust::device_vector<FeatureInstance> firstLevel(5);
	thrust::device_vector<FeatureInstance> secondLevel(5);

	writeFirstTwoLevels <<< insertGrid, 256 >>> (
		instanceMapResult->map->getBean()
		, thrust::raw_pointer_cast(cliques.data())
		, groupNumber.data()
		, itemNumber.data()
		, pairsA.data()
		, pairsB.data()
		, 5
		, firstLevel.data()
		, secondLevel.data()
		);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	std::vector<FeatureInstance> expectedFirstLevel;
	{
		FeatureInstance fi;

		fi.field = 0x000A0001;
		expectedFirstLevel.push_back(fi);

		fi.field = 0x000A0002;
		expectedFirstLevel.push_back(fi);

		fi.field = 0x000B0001;
		expectedFirstLevel.push_back(fi);

		fi.field = 0x000B0002;
		expectedFirstLevel.push_back(fi);

		fi.field = 0x000B0002;
		expectedFirstLevel.push_back(fi);
	}

	std::vector<FeatureInstance> expectedSecondLevel;
	{
		FeatureInstance fi;

		//a1 - b1
		//a2 - b2
		//a2 - c2
		//b1 - c1
		//b2 - c1
		//b2 - c3		

		fi.field = 0x000B0001;
		expectedSecondLevel.push_back(fi);

		fi.field = 0x000B0002;
		expectedSecondLevel.push_back(fi);

		fi.field = 0x000C0001;
		expectedSecondLevel.push_back(fi);

		fi.field = 0x000C0001;
		expectedSecondLevel.push_back(fi);

		fi.field = 0x000C0003;
		expectedSecondLevel.push_back(fi);
	}
	

	thrust::host_vector<FeatureInstance> resultFirstLevel = firstLevel;
	thrust::host_vector<FeatureInstance> resultSecondLevel = secondLevel;

	REQUIRE(std::equal(expectedFirstLevel.begin(), expectedFirstLevel.end(), resultFirstLevel.begin()));
	REQUIRE(std::equal(expectedSecondLevel.begin(), expectedSecondLevel.end(), resultSecondLevel.begin()));
}
//--------------------------------------------------------------


/*
Test for graph

		 C3-D2
		  |
A1-B1-C1-B2-A2-C2-D1
   
*/
TEST_CASE_METHOD(BaseCudaTestHandler, "Instance tree helpers | insert third level counts")
{
	thrust::device_vector<FeatureInstance> pairsA;
	thrust::device_vector<FeatureInstance> pairsB;
	{
		/*
			a1 - b1
			a2 - b2
			a2 - c2

			b1 - c1
			b2 - c1
			b2 - c3

			c2 - d1
			c3 - d2

			a1-b1-c1  1
				  
			a2-b2 c1  2
				  c3

			b1-c1-	  0
			b2-c1-    0

			b2-c3-d2  1


		*/
		std::vector<FeatureInstance> hPairsA = {
			 { 0x000A0001 }
			,{ 0x000A0002 }
			,{ 0x000A0002 }

			,{ 0x000B0001 }
			,{ 0x000B0002 }
			,{ 0x000B0002 }

			,{ 0x000C0002 }
			,{ 0x000C0003 }
		};

		std::vector<FeatureInstance> hPairsB = {
			 { 0x000B0001 }
			,{ 0x000B0002 }
			,{ 0x000C0002 }

			,{ 0x000C0001 }
			,{ 0x000C0001 }
			,{ 0x000C0003 }

			,{ 0x000D0001 }
			,{ 0x000D0002 }
		};

		pairsA = hPairsA;
		pairsB = hPairsB;
	}

	auto instanceNeighboursMap = InstanceTypedNeighboursMapCreator::createTypedNeighboursListMap(
		pairsA
		, pairsB
	);

	thrust::device_vector<thrust::device_vector<unsigned short>> cliquesData;
	thrust::host_vector<thrust::device_vector<unsigned short>> hcliquesData;
	{

		std::vector<unsigned short> first = { 0x000A, 0x000B, 0x000C };
		std::vector<unsigned short> second = { 0x000B, 0x000C, 0x000D };

		hcliquesData.push_back(first);
		hcliquesData.push_back(second);

		cliquesData = hcliquesData;
	}

	thrust::device_vector<thrust::device_ptr<const unsigned short>> cliques;
	{
		std::vector<thrust::device_ptr<const unsigned short>> hcliques;
		for (const thrust::device_vector<unsigned short>& vec : hcliquesData)
			hcliques.push_back(vec.data());

		cliques = hcliques;
	}

	thrust::device_vector<FeatureInstance> secondLevelInstances;
	{
		std::vector<FeatureInstance> expectedSecondLevel;
		{
			FeatureInstance fi;

			/*
			a1 - b1
			a2 - b2
			a2 - c2
			b1 - c1
			b2 - c1
			b2 - c3
			*/

			fi.field = 0x000B0001;
			expectedSecondLevel.push_back(fi);

			fi.field = 0x000B0002;
			expectedSecondLevel.push_back(fi);

			fi.field = 0x000C0001;
			expectedSecondLevel.push_back(fi);

			fi.field = 0x000C0001;
			expectedSecondLevel.push_back(fi);

			fi.field = 0x000C0003;
			expectedSecondLevel.push_back(fi);
		}

		secondLevelInstances = expectedSecondLevel;
	}


	thrust::device_vector<unsigned int> group;
	thrust::device_vector<thrust::device_ptr<unsigned int>> groups;
	{
		//        g   i 
 		//a1 - b1   0   0
		//a2 - b2   0   1
		//a2 - c2   1   0
		//b1 - c1   2   0   
		//b2 - c1   2   1
		//b2 - c3   2   2
		

		std::vector<unsigned int> hgroups = { 0, 0, 1, 1, 1 };

		group = hgroups;
	}

	groups.push_back(group.data());
	groups.push_back(group.data());

	// ####################################################################

	const unsigned int outpuCount = secondLevelInstances.size();

	thrust::device_vector<unsigned int> thirdLevelCounts(outpuCount);

	thrust::device_vector<unsigned int> result(outpuCount);

	dim3 insertGrid;
	findSmallest2D(outpuCount, 256, insertGrid.x, insertGrid.y);

	thrust::device_vector<bool> integrityMask(outpuCount, true);

	fillWithNextLevelCountsFromTypedNeighbour <<< insertGrid, 256 >>> (
		instanceNeighboursMap->map->getBean()
		, thrust::raw_pointer_cast(cliques.data())
		, thrust::raw_pointer_cast(groups.data())
		, secondLevelInstances.data()
		, outpuCount
		, 2
		, integrityMask.data()
		, result.data()
	);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	std::vector<unsigned int> expectedCounts{ 1, 2, 0, 0, 1 };

	thrust::host_vector<unsigned int> resultCounts = result;

	REQUIRE(std::equal(expectedCounts.begin(), expectedCounts.end(), resultCounts.begin()));
}



/*
Test for graph

		 C3-D2
		 |
A1-B1-C1-B2-A2-C2-D1


a1-b1-c1  1

a2-b2 c1  2
	  c3

b1-c1-	  0
b2-c1-    0

b2-c3-d2  1
*/
TEST_CASE_METHOD(BaseCudaTestHandler, "Instance tree helpers | insert third..n level instances")
{
	// planessweep data
	thrust::device_vector<FeatureInstance> pairsA;
	thrust::device_vector<FeatureInstance> pairsB;
	{
		/*
		a1 - b1
		a2 - b2
		a2 - c2
		b1 - c1
		b2 - c1
		b2 - c3
		c2 - d1
		c3 - d2
		*/
		std::vector<FeatureInstance> hPairsA = {
			{ 0x000A0001 }
			,{ 0x000A0002 }
			,{ 0x000A0002 }
			,{ 0x000B0001 }
			,{ 0x000B0002 }
			,{ 0x000B0002 }
			,{ 0x000C0002 }
			,{ 0x000C0003 }
		};

		std::vector<FeatureInstance> hPairsB = {
			{ 0x000B0001 }
			,{ 0x000B0002 }
			,{ 0x000C0002 }
			,{ 0x000C0001 }
			,{ 0x000C0001 }
			,{ 0x000C0003 }
			,{ 0x000D0001 }
			,{ 0x000D0002 }
		};

		pairsA = hPairsA;
		pairsB = hPairsB;
	}


	// instance neighbour map
	auto instanceNeighboursMap = InstanceTypedNeighboursMapCreator::createTypedNeighboursListMap(
		pairsA
		, pairsB
	);

	// clique data
	thrust::device_vector<thrust::device_vector<unsigned short>> cliquesData;
	thrust::host_vector<thrust::device_vector<unsigned short>> hcliquesData;
	{

		std::vector<unsigned short> first = { 0x000A, 0x000B, 0x000C };
		std::vector<unsigned short> second = { 0x000B, 0x000C, 0x000D };

		hcliquesData.push_back(first);
		hcliquesData.push_back(second);

		cliquesData = hcliquesData;
	}

	thrust::device_vector<thrust::device_ptr<const unsigned short>> cliques;
	{
		std::vector<thrust::device_ptr<const unsigned short>> hcliques;
		for (const thrust::device_vector<unsigned short>& vec : hcliquesData)
			hcliques.push_back(vec.data());

		cliques = hcliques;
	}


	// forgroups result

	std::vector<UIntThrustVectorPtr> forGroupGroups;
	thrust::device_vector<thrust::device_ptr<unsigned int>> forGroupGroupsDevPtrs;
	{
		std::vector<thrust::device_ptr<unsigned int>> tempDevPtr;

		std::vector<unsigned int> hFirstLevelGroups = { 0, 0, 1, 1, 1 };

		forGroupGroups.push_back(std::make_shared<UIntThrustVector>(hFirstLevelGroups));
		tempDevPtr.push_back(forGroupGroups.back()->data());

		std::vector<unsigned int> hsecondLevelGroup = { 0, 0, 1, 1, 1 };

		forGroupGroups.push_back(std::make_shared<UIntThrustVector>(hsecondLevelGroup));
		tempDevPtr.push_back(forGroupGroups.back()->data());

		/*
			groups count 
			1, 2, 0, 0, 1
		*/
		std::vector<unsigned int> hthirdLevel = { 0, 1, 1, 4 };

		forGroupGroups.push_back(std::make_shared<UIntThrustVector>(hthirdLevel));
		tempDevPtr.push_back(forGroupGroups.back()->data());

		forGroupGroupsDevPtrs = tempDevPtr;
	}

	thrust::device_vector<unsigned int> itemsNumber;
	{
		std::vector<unsigned int> hItemumbers = { 0, 0, 1, 0 };
		itemsNumber = hItemumbers;
	}

	// last ready level instances
	thrust::device_vector<FeatureInstance> secondLevelInstances;
	{
		std::vector<FeatureInstance> hSecondLevelInstances;
		{
			FeatureInstance fi;

			fi.field = 0x000B0001;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000B0002;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000C0001;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000C0001;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000C0003;
			hSecondLevelInstances.push_back(fi);
		}

		secondLevelInstances = hSecondLevelInstances;
	}


	// ####################################################################
	
	const unsigned int outpuCount = forGroupGroups.back()->size();

	thrust::device_vector<FeatureInstance> result(outpuCount);

	dim3 insertGrid;
	findSmallest2D(outpuCount, 256, insertGrid.x, insertGrid.y);
	
	fillLevelInstancesFromNeighboursList << < insertGrid, 256 >> > (
		instanceNeighboursMap->map->getBean()
		, cliques.data().get()
		, forGroupGroupsDevPtrs.data().get()
		, itemsNumber.data()
		, secondLevelInstances.data()
		, pairsB.data()
		, outpuCount
		, 2
		, result.data()
		);
		
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	std::vector<FeatureInstance> expectedThirdLevelInstances;
	{ 
		FeatureInstance fi;
		/*
		a1-b1-c1  1

		a2-b2 c1  2
			  c3

		b1-c1-	  0
		b2-c1-    0

		b2-c3-d2  1
		*/

		fi.field = 0x000C0001;
		expectedThirdLevelInstances.push_back(fi);

		fi.field = 0x000C0001;
		expectedThirdLevelInstances.push_back(fi);

		fi.field = 0x000C0003;
		expectedThirdLevelInstances.push_back(fi);

		fi.field = 0x000D0002;
		expectedThirdLevelInstances.push_back(fi);
	}

	thrust::host_vector<FeatureInstance> calculatedThirdLevelInstances = result;

	REQUIRE(std::equal(expectedThirdLevelInstances.begin(), expectedThirdLevelInstances.end(), calculatedThirdLevelInstances.begin()));
}
// -------------------------------------------------------------------------------------------------------------------------------

/*
Test for graph

		 C3-D2
		 | /
A1-B1-C1-B2-A2-C2-D1

*/
TEST_CASE_METHOD(BaseCudaTestHandler, "Instance tree helpers | check clique integrity")
{
	// planessweep data
	thrust::device_vector<FeatureInstance> pairsA;
	thrust::device_vector<FeatureInstance> pairsB;
	{
		/*
		a1 - b1
		a2 - b2
		a2 - c2
		b1 - c1
		b2 - c1
		b2 - c3
		b2 - d2
		c2 - d1
		c3 - d2
		*/
		std::vector<FeatureInstance> hPairsA = {
			{ 0x000A0001 }
			,{ 0x000A0002 }
			,{ 0x000A0002 }
			,{ 0x000B0001 }
			,{ 0x000B0002 }
			,{ 0x000B0002 }
			,{ 0x000B0002 }
			,{ 0x000C0002 }
			,{ 0x000C0003 }
		};

		std::vector<FeatureInstance> hPairsB = {
			{ 0x000B0001 }
			,{ 0x000B0002 }
			,{ 0x000C0002 }
			,{ 0x000C0001 }
			,{ 0x000C0001 }
			,{ 0x000C0003 }
			,{ 0x000D0002 }
			,{ 0x000D0001 }
			,{ 0x000D0002 }
		};

		pairsA = hPairsA;
		pairsB = hPairsB;
	}


	// instance neighbour map
	auto instanceNeighboursMap = InstanceTypedNeighboursMapCreator::createTypedNeighboursListMap(
		pairsA
		, pairsB
	);

	// clique data
	thrust::device_vector<thrust::device_vector<unsigned short>> cliquesData;
	thrust::host_vector<thrust::device_vector<unsigned short>> hcliquesData;
	{

		std::vector<unsigned short> first = { 0x000A, 0x000B, 0x000C };
		std::vector<unsigned short> second = { 0x000B, 0x000C, 0x000D };

		hcliquesData.push_back(first);
		hcliquesData.push_back(second);

		cliquesData = hcliquesData;
	}

	thrust::device_vector<thrust::device_ptr<const unsigned short>> cliques;
	{
		std::vector<thrust::device_ptr<const unsigned short>> hcliques;
		for (const thrust::device_vector<unsigned short>& vec : hcliquesData)
			hcliques.push_back(vec.data());

		cliques = hcliques;
	}


	// forgroups result

	std::vector<UIntThrustVectorPtr> forGroupGroups;
	thrust::device_vector<thrust::device_ptr<unsigned int>> forGroupGroupsDevPtrs;
	{
		std::vector<thrust::device_ptr<unsigned int>> tempDevPtr;

		std::vector<unsigned int> hFirstLevelGroups = { 0, 0, 1, 1, 1 };

		forGroupGroups.push_back(std::make_shared<UIntThrustVector>(hFirstLevelGroups));
		tempDevPtr.push_back(forGroupGroups.back()->data());

		std::vector<unsigned int> hsecondLevelGroup = { 0, 0, 1, 1, 1 };

		forGroupGroups.push_back(std::make_shared<UIntThrustVector>(hsecondLevelGroup));
		tempDevPtr.push_back(forGroupGroups.back()->data());

		/*
		groups count
		1, 2, 0, 0, 1
		*/
		std::vector<unsigned int> hthirdLevel = { 0, 1, 1, 4 };

		forGroupGroups.push_back(std::make_shared<UIntThrustVector>(hthirdLevel));
		tempDevPtr.push_back(forGroupGroups.back()->data());

		forGroupGroupsDevPtrs = tempDevPtr;
	}

	thrust::device_vector<unsigned int> itemsNumber;
	{
		std::vector<unsigned int> hItemumbers = { 0, 0, 1, 0 };
		itemsNumber = hItemumbers;
	}



	// instances levels
	thrust::device_vector<FeatureInstance> firstLevelInstances;
	{
		/*
		a1-b1-c1  1

		a2-b2 c1  2
			  c3

		b1-c1-	  0
		b2-c1-    0

		b2-c3-d2  1
		*/

		std::vector<FeatureInstance> hFirstLevelInstances;
		{
			FeatureInstance fi;

			fi.field = 0x000A0001;
			hFirstLevelInstances.push_back(fi);

			fi.field = 0x000A0002;
			hFirstLevelInstances.push_back(fi);

			fi.field = 0x000B0001;
			hFirstLevelInstances.push_back(fi);

			fi.field = 0x000B0002;
			hFirstLevelInstances.push_back(fi);

			fi.field = 0x000B0002;
			hFirstLevelInstances.push_back(fi);
		}

		firstLevelInstances = hFirstLevelInstances;
	}

	thrust::device_vector<FeatureInstance> secondLevelInstances;
	{
		std::vector<FeatureInstance> hSecondLevelInstances;
		{
			FeatureInstance fi;

			fi.field = 0x000B0001;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000B0002;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000C0001;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000C0001;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000C0003;
			hSecondLevelInstances.push_back(fi);
		}

		secondLevelInstances = hSecondLevelInstances;
	}

	thrust::device_vector<FeatureInstance> thirdLevelInstances;
	{
		std::vector<FeatureInstance> hThirdLevelInstances;
		{
			FeatureInstance fi;

			fi.field = 0x000C0001;
			hThirdLevelInstances.push_back(fi);

			fi.field = 0x000C0001;
			hThirdLevelInstances.push_back(fi);

			fi.field = 0x000C0003;
			hThirdLevelInstances.push_back(fi);

			fi.field = 0x000D0002;
			hThirdLevelInstances.push_back(fi);
		}

		thirdLevelInstances = hThirdLevelInstances;
	}

	thrust::device_vector<thrust::device_ptr<FeatureInstance>> instancesOnLevels;
	{
		std::vector<thrust::device_ptr<FeatureInstance>> hInstancesOnLevels;

		hInstancesOnLevels.push_back(firstLevelInstances.data());
		hInstancesOnLevels.push_back(secondLevelInstances.data());
		hInstancesOnLevels.push_back(thirdLevelInstances.data());

		instancesOnLevels = hInstancesOnLevels;
	}

	const unsigned int outpuCount = forGroupGroups.back()->size();

	thrust::device_vector<bool> result(outpuCount);

	dim3 insertGrid;
	findSmallest2D(outpuCount, 256, insertGrid.x, insertGrid.y);

	markAsPartOfCurrentCliqueInstance <<< insertGrid, 256 >>> (
		instanceNeighboursMap->map->getBean()
		, forGroupGroupsDevPtrs.data().get()
		, instancesOnLevels.data().get()
		, thirdLevelInstances.data()
		, pairsB.data()
		, outpuCount
		, 2
		, result.data()
	);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	std::vector<bool> expected = { false, false, false, true };
	thrust::host_vector<bool> calculated = result;

	REQUIRE(std::equal(expected.begin(), expected.end(), calculated.begin()));
}
// ---------------------------------------------------------------------------------------------


TEST_CASE_METHOD(BaseCudaTestHandler, "Instance tree helpers | reverese generate simple 1")
{
	// instances levels
	thrust::device_vector<FeatureInstance> firstLevelInstances;
	{
		std::vector<FeatureInstance> hFirstLevelInstances;
		{
			FeatureInstance fi;

			fi.field = 0x000A0001; 
			hFirstLevelInstances.push_back(fi);

			fi.field = 0x000A0002;
			hFirstLevelInstances.push_back(fi);

			fi.field = 0x000B0001;
			hFirstLevelInstances.push_back(fi);

			fi.field = 0x000B0002;
			hFirstLevelInstances.push_back(fi);

			fi.field = 0x000B0002;
			hFirstLevelInstances.push_back(fi);
		}

		firstLevelInstances = hFirstLevelInstances;
	}

	thrust::device_vector<FeatureInstance> secondLevelInstances;
	{
		std::vector<FeatureInstance> hSecondLevelInstances;
		{
			FeatureInstance fi;

			fi.field = 0x000B0001;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000B0002;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000C0001;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000C0001;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000C0003;
			hSecondLevelInstances.push_back(fi);
		}

		secondLevelInstances = hSecondLevelInstances;
	}

	thrust::device_vector<FeatureInstance> thirdLevelInstances;
	{
		std::vector<FeatureInstance> hThirdLevelInstances;
		{
			FeatureInstance fi;

			fi.field = 0x000C0001;
			hThirdLevelInstances.push_back(fi);

			fi.field = 0x000C0001;
			hThirdLevelInstances.push_back(fi);

			fi.field = 0x000C0003;
			hThirdLevelInstances.push_back(fi);

			fi.field = 0x000D0002;
			hThirdLevelInstances.push_back(fi);
		}

		thirdLevelInstances = hThirdLevelInstances;
	}

	thrust::device_vector<thrust::device_ptr<FeatureInstance>> instancesOnLevels;
	{
		std::vector<thrust::device_ptr<FeatureInstance>> hInstancesOnLevels;

		hInstancesOnLevels.push_back(firstLevelInstances.data());
		hInstancesOnLevels.push_back(secondLevelInstances.data());
		hInstancesOnLevels.push_back(thirdLevelInstances.data());

		instancesOnLevels = hInstancesOnLevels;
	}


	// forgroups result

	std::vector<UIntThrustVectorPtr> forGroupGroups;
	thrust::device_vector<thrust::device_ptr<unsigned int>> forGroupGroupsDevPtrs;
	{
		std::vector<thrust::device_ptr<unsigned int>> tempDevPtr;

		std::vector<unsigned int> hFirstLevelGroups = { 0, 0, 1, 1, 1 };

		forGroupGroups.push_back(std::make_shared<UIntThrustVector>(hFirstLevelGroups));
		tempDevPtr.push_back(forGroupGroups.back()->data());

		std::vector<unsigned int> hsecondLevelGroup = { 0, 0, 1, 1, 1 };

		forGroupGroups.push_back(std::make_shared<UIntThrustVector>(hsecondLevelGroup));
		tempDevPtr.push_back(forGroupGroups.back()->data());

		/*
		groups count
		1, 2, 0, 0, 1
		*/
		std::vector<unsigned int> hthirdLevel = { 0, 1, 1, 4 };

		forGroupGroups.push_back(std::make_shared<UIntThrustVector>(hthirdLevel));
		tempDevPtr.push_back(forGroupGroups.back()->data());

		forGroupGroupsDevPtrs = tempDevPtr;
	}
	
	// write positions
	thrust::device_vector<unsigned int> writePositions;
	{
		std::vector<unsigned int> hWritePositions = { 0, 1, 1, 2 };
		writePositions = hWritePositions;			
	}

	// integrity mask
	thrust::device_vector<bool> integrityMask;
	{
		std::vector<bool> hIntegrityMask = { true, true, false, true };
		integrityMask = hIntegrityMask;
	}

	// result
	
	const unsigned int endCount = 4;
	const unsigned int consistentCount = 3;
	const unsigned int cliqueSize = 3;

	thrust::device_vector<FeatureInstance> result(consistentCount * cliqueSize);

	dim3 insertGrid;
	findSmallest2D(endCount, 256, insertGrid.x, insertGrid.y);

	reverseGenerateCliquesInstances <<< insertGrid, 256 >>> (
		forGroupGroupsDevPtrs.data().get()
		, instancesOnLevels.data().get()
		, endCount
		, consistentCount
		, cliqueSize
		, integrityMask.data()
		, writePositions.data()
		, result.data()
	);


	/*
	a1-b1-c1  1

	a2-b2 c1  2
		  c3

	b1-c1-	  0
	b2-c1-    0

	b2-c3-d2  1
	*/

	std::vector<FeatureInstance> expected = {
		 { 0x000A0001 }, { 0x000A0002 }, { 0x000B0002 }
		,{ 0x000B0001 }, { 0x000B0002 }, { 0x000C0003 }
		,{ 0x000C0001 }, { 0x000C0001 }, { 0x000D0002 }
	};

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	thrust::host_vector<FeatureInstance> copmuted = result;

	REQUIRE(std::equal(expected.begin(), expected.end(), copmuted.begin()));
}


TEST_CASE_METHOD(BaseCudaTestHandler, "Instance tree helpers | reverese generate simple 2")
{
	// instances levels
	thrust::device_vector<FeatureInstance> firstLevelInstances;
	{
		std::vector<FeatureInstance> hFirstLevelInstances;
		{
			FeatureInstance fi;

			fi.field = 0x000A0001;
			hFirstLevelInstances.push_back(fi);

			fi.field = 0x000A0002;
			hFirstLevelInstances.push_back(fi);

			fi.field = 0x000B0001;
			hFirstLevelInstances.push_back(fi);

			fi.field = 0x000B0002;
			hFirstLevelInstances.push_back(fi);

			fi.field = 0x000B0002;
			hFirstLevelInstances.push_back(fi);
		}

		firstLevelInstances = hFirstLevelInstances;
	}

	thrust::device_vector<FeatureInstance> secondLevelInstances;
	{
		std::vector<FeatureInstance> hSecondLevelInstances;
		{
			FeatureInstance fi;

			fi.field = 0x000B0001;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000B0002;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000C0001;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000C0001;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000C0003;
			hSecondLevelInstances.push_back(fi);
		}

		secondLevelInstances = hSecondLevelInstances;
	}

	thrust::device_vector<FeatureInstance> thirdLevelInstances;
	{
		std::vector<FeatureInstance> hThirdLevelInstances;
		{
			FeatureInstance fi;

			fi.field = 0x000C0001;
			hThirdLevelInstances.push_back(fi);

			fi.field = 0x000C0001;
			hThirdLevelInstances.push_back(fi);

			fi.field = 0x000C0003;
			hThirdLevelInstances.push_back(fi);

			fi.field = 0x000D0002;
			hThirdLevelInstances.push_back(fi);
		}

		thirdLevelInstances = hThirdLevelInstances;
	}

	thrust::device_vector<thrust::device_ptr<FeatureInstance>> instancesOnLevels;
	{
		std::vector<thrust::device_ptr<FeatureInstance>> hInstancesOnLevels;

		hInstancesOnLevels.push_back(firstLevelInstances.data());
		hInstancesOnLevels.push_back(secondLevelInstances.data());
		hInstancesOnLevels.push_back(thirdLevelInstances.data());

		instancesOnLevels = hInstancesOnLevels;
	}


	// forgroups result

	std::vector<UIntThrustVectorPtr> forGroupGroups;
	thrust::device_vector<thrust::device_ptr<unsigned int>> forGroupGroupsDevPtrs;
	{
		std::vector<thrust::device_ptr<unsigned int>> tempDevPtr;

		std::vector<unsigned int> hFirstLevelGroups = { 0, 0, 1, 1, 1 };

		forGroupGroups.push_back(std::make_shared<UIntThrustVector>(hFirstLevelGroups));
		tempDevPtr.push_back(forGroupGroups.back()->data());

		std::vector<unsigned int> hsecondLevelGroup = { 0, 0, 1, 1, 1 };

		forGroupGroups.push_back(std::make_shared<UIntThrustVector>(hsecondLevelGroup));
		tempDevPtr.push_back(forGroupGroups.back()->data());

		/*
		groups count
		1, 2, 0, 0, 1
		*/
		std::vector<unsigned int> hthirdLevel = { 0, 1, 1, 4 };

		forGroupGroups.push_back(std::make_shared<UIntThrustVector>(hthirdLevel));
		tempDevPtr.push_back(forGroupGroups.back()->data());

		forGroupGroupsDevPtrs = tempDevPtr;
	}

	// write positions
	thrust::device_vector<unsigned int> writePositions;
	{
		std::vector<unsigned int> hWritePositions = { 0, 1, 1, 2 };
		writePositions = hWritePositions;
	}

	// integrity mask
	thrust::device_vector<bool> integrityMask;
	{
		std::vector<bool> hIntegrityMask = { true, false, true, true };
		integrityMask = hIntegrityMask;
	}

	// result

	const unsigned int endCount = 4;
	const unsigned int consistentCount = 3;
	const unsigned int cliqueSize = 3;

	thrust::device_vector<FeatureInstance> result(consistentCount * cliqueSize);

	dim3 insertGrid;
	findSmallest2D(endCount, 256, insertGrid.x, insertGrid.y);

	reverseGenerateCliquesInstances << < insertGrid, 256 >> > (
		forGroupGroupsDevPtrs.data().get()
		, instancesOnLevels.data().get()
		, endCount
		, consistentCount
		, cliqueSize
		, integrityMask.data()
		, writePositions.data()
		, result.data()
		);


	/*
	a1-b1-c1  1

	a2-b2 c1  2
	      c3

	b1-c1-	  0
	b2-c1-    0

	b2-c3-d2  1
	*/

	std::vector<FeatureInstance> expected = {
		{ 0x000A0001 },{ 0x000A0002 },{ 0x000B0002 }
		,{ 0x000B0001 },{ 0x000B0002 },{ 0x000C0003 }
		,{ 0x000C0001 },{ 0x000C0003 },{ 0x000D0002 }
	};

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	thrust::host_vector<FeatureInstance> copmuted = result;

	REQUIRE(std::equal(expected.begin(), expected.end(), copmuted.begin()));
}

TEST_CASE_METHOD(BaseCudaTestHandler, "Instance tree helpers | reverese generate simple 2-size")
{
	// instances levels
	thrust::device_vector<FeatureInstance> firstLevelInstances;
	{
		std::vector<FeatureInstance> hFirstLevelInstances;
		{
			FeatureInstance fi;

			fi.field = 0x000A0001;
			hFirstLevelInstances.push_back(fi);

			fi.field = 0x000A0002;
			hFirstLevelInstances.push_back(fi);

			fi.field = 0x000B0001;
			hFirstLevelInstances.push_back(fi);

			fi.field = 0x000B0002;
			hFirstLevelInstances.push_back(fi);

			fi.field = 0x000B0002;
			hFirstLevelInstances.push_back(fi);
		}

		firstLevelInstances = hFirstLevelInstances;
	}

	thrust::device_vector<FeatureInstance> secondLevelInstances;
	{
		std::vector<FeatureInstance> hSecondLevelInstances;
		{
			FeatureInstance fi;

			fi.field = 0x000B0001;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000B0002;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000C0001;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000C0001;
			hSecondLevelInstances.push_back(fi);

			fi.field = 0x000C0003;
			hSecondLevelInstances.push_back(fi);
		}

		secondLevelInstances = hSecondLevelInstances;
	}


	thrust::device_vector<thrust::device_ptr<FeatureInstance>> instancesOnLevels;
	{
		std::vector<thrust::device_ptr<FeatureInstance>> hInstancesOnLevels;

		hInstancesOnLevels.push_back(firstLevelInstances.data());
		hInstancesOnLevels.push_back(secondLevelInstances.data());

		instancesOnLevels = hInstancesOnLevels;
	}


	// forgroups result

	std::vector<UIntThrustVectorPtr> forGroupGroups;
	thrust::device_vector<thrust::device_ptr<unsigned int>> forGroupGroupsDevPtrs;
	{
		std::vector<thrust::device_ptr<unsigned int>> tempDevPtr;

		std::vector<unsigned int> hFirstLevelGroups = { 0, 0, 1, 1, 1 };

		forGroupGroups.push_back(std::make_shared<UIntThrustVector>(hFirstLevelGroups));
		tempDevPtr.push_back(forGroupGroups.back()->data());

		std::vector<unsigned int> hsecondLevelGroup = { 0, 0, 1, 1, 1 };

		forGroupGroups.push_back(std::make_shared<UIntThrustVector>(hsecondLevelGroup));
		tempDevPtr.push_back(forGroupGroups.back()->data());

		forGroupGroupsDevPtrs = tempDevPtr;
	}

	// write positions
	thrust::device_vector<unsigned int> writePositions;
	{
		std::vector<unsigned int> hWritePositions = { 0, 1, 1, 2, 3 };
		writePositions = hWritePositions;
	}

	// integrity mask
	thrust::device_vector<bool> integrityMask;
	{
		std::vector<bool> hIntegrityMask = { true, false, true, true, true };
		integrityMask = hIntegrityMask;
	}

	// result

	const unsigned int endCount = 5;
	const unsigned int consistentCount = 4;
	const unsigned int cliqueSize = 2;

	thrust::device_vector<FeatureInstance> result(consistentCount * cliqueSize);

	dim3 insertGrid;
	findSmallest2D(endCount, 256, insertGrid.x, insertGrid.y);

	reverseGenerateCliquesInstances << < insertGrid, 256 >> > (
		forGroupGroupsDevPtrs.data().get()
		, instancesOnLevels.data().get()
		, endCount
		, consistentCount
		, cliqueSize
		, integrityMask.data()
		, writePositions.data()
		, result.data()
		);


	/*
	a1-b1  true
					a2-b2  false
	b1-c1  true
	b2-c1  true
	b2-c3  true
	*/

	std::vector<FeatureInstance> expected = {
		 { 0x000A0001 },{ 0x000B0001 },{ 0x000B0002 },{ 0x000B0002 }
		,{ 0x000B0001 },{ 0x000C0001 },{ 0x000C0001 },{ 0x000C0003 }
	};

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	thrust::host_vector<FeatureInstance> copmuted = result;

	REQUIRE(std::equal(expected.begin(), expected.end(), copmuted.begin()));
}

TEST_CASE_METHOD(BaseCudaTestHandler, "Instance tree helpers | generate write positions")
{
	thrust::device_vector<bool> integrityMask;
	{
		std::vector<bool> hIntegrityMask{ true, true, true, false, false, true };
		integrityMask = hIntegrityMask;
	}

	thrust::device_vector<unsigned int> writePositions(6);

	auto consistentCount = fillWritePositionsAndReturnCount(
		integrityMask
		, writePositions
		, 6
	);

	std::vector<unsigned int> expected = { 0, 1, 2, 3, 3, 3 };

	thrust::host_vector<unsigned int> calculated = writePositions;

	REQUIRE(consistentCount == 4);
	REQUIRE(std::equal(expected.begin(), expected.end(), calculated.begin()));
}


TEST_CASE_METHOD(BaseCudaTestHandler, "Instance tree helpers | generate write positions, redundant integrity mask")
{
	thrust::device_vector<bool> integrityMask;
	{
		std::vector<bool> hIntegrityMask{ true, true, true, false, false, true, false, false, true };
		integrityMask = hIntegrityMask;
	}

	thrust::device_vector<unsigned int> writePositions(6);

	auto consistentCount = fillWritePositionsAndReturnCount(
		integrityMask
		, writePositions
		, 6
	);

	std::vector<unsigned int> expected = { 0, 1, 2, 3, 3, 3 };

	thrust::host_vector<unsigned int> calculated = writePositions;

	REQUIRE(consistentCount == 4);
	REQUIRE(std::equal(expected.begin(), expected.end(), calculated.begin()));
}