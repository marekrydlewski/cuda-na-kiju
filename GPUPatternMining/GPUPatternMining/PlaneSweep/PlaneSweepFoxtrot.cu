#include "hip/hip_runtime.h"
#include "PlaneSweepFoxtrot.h"

#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "../Common/MiningCommon.h"

#include "../Entities/NeighboursListInfoHolder.h"
#include "../Common/CommonOperations.h"

#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>

namespace PlaneSweep
{
	namespace Foxtrot
	{
		__global__ void countNeighbours(
			float* xCoords
			, float* yCoords
			, FeatureInstance* instances
			, int count
			, float radius
			, float radiusSquared
			, int warpsCount
			, UInt* resultNeighboursCount)
		{
			// btid
			int blockThreadId = threadIdx.x;
			//gid
			int globalId = computeLinearAddressFrom2D();
			// wid
			int warpId = globalId / 32;
			// bwid
			int blockWarpId = blockThreadId / 32;
			// wtid
			int warpThreadId = threadIdx.x % 32;

			__shared__ volatile bool * flags;
			__shared__ volatile UInt * found;

			if (threadIdx.x == 0)
			{
				flags = static_cast<bool*>(malloc((blockDim.x / 32) * sizeof(bool)));
				found = static_cast<UInt*>(malloc(blockDim.x * uintSize));
			}

			__syncthreads();

			//uint start= wid        * ((inSize-1 ) / warpCount ) + max(0, - warpCount  + wid          + (inSize - 1) % warpCount ) + 1;
			int start = warpId * ((count - 1) / warpsCount) + max(0, -warpsCount + warpId + (count - 1) % warpsCount) + 1;
			//uint stop=(wid         + 1) * ((inSize-1 ) / warpCount ) + max(0, -warpCount  +  (inSize- 1) % warpCount  + wid          + 1);
			int stop = (warpId + 1) * ((count - 1) / warpsCount) + max(0, -warpsCount + warpId + 1 + (count - 1) % warpsCount);

			if (warpId == 63)
			{
				found[blockThreadId] = 0;
			}

			if (warpId < warpsCount)
			{
				found[blockThreadId] = 0;

				for (UInt i = start; i <= stop; i++)
				{
					float px = xCoords[i];
					float py = yCoords[i];

					flags[blockWarpId] = false;

					for (int j = i - 32; j >= -32; j -= 32)
					{
						int localId = warpThreadId + j;

						if (localId >= 0)
						{
							if (instances[i].fields.featureId == instances[localId].fields.featureId)
								continue;

							float lx = xCoords[localId];

							if ((px - lx) > radius)
							{
								flags[blockWarpId] = true;
							}

							float ly = yCoords[localId];

							if ((MiningCommon::distance(px, py, lx, ly) <= radiusSquared))
							{
								found[blockThreadId] += 1;
							}
						}

						if (flags[blockWarpId])
						{
							break;
						}
					}
				}

				MiningCommon::intraWarpReduce(found);

				// warpThreadId == 0?
				if (globalId == warpId * 32)
				{
					resultNeighboursCount[warpId] = found[blockWarpId * 32 + 31];
				}
			}

			__syncthreads();

			if (threadIdx.x == 0)
			{
				free(const_cast<bool*>(flags));
				free(const_cast<UInt*>(found));
			}
		}
		// --------------------------------------------------------------------------------------------------------------------------------------

		__global__ void findNeighbours(
			float* xCoords
			, float* yCoords
			, FeatureInstance* instances
			, int count
			, float radius
			, float radiusSquared
			, int warpsCount
			, UInt *outStarts
			, FeatureInstance* out_a
			, FeatureInstance* out_b)
		{
			// btid
			int blockThreadId = threadIdx.x;
			//gid
			int globalId = computeLinearAddressFrom2D();
			// wid
			int warpId = globalId / 32;
			// bwid
			int blockWarpId = blockThreadId / 32;
			// wtid
			int warpThreadId = threadIdx.x % 32;

			// const UInt underBuffId = blockWarpId * 64 + warpThreadId;
			// const int aboveBuffId = blockWarpId * 64 + 32 + warpThreadId;

			__shared__ volatile UInt* scanBuf;
			__shared__ volatile bool* flags;
			__shared__ volatile bool* found;
			volatile __shared__ UInt* buffA;
			volatile __shared__ UInt* buffB;
			__shared__ UInt* warpBuffPos;


			FeatureInstance temp_a;
			FeatureInstance temp_b;
			// UInt localStart = 0;

			if (threadIdx.x == 0)
			{
				// check Allocating http://www.drdobbs.com/parallel/a-massively-parallel-stack-for-data-allo/240162018?pgno=1
				// measure dynamic allocating in different warps
				scanBuf = static_cast<UInt*>(malloc(blockDim.x * uintSize));
				flags = static_cast<bool*>(malloc((blockDim.x / 32) * sizeof(bool)));
				found = static_cast<bool*>(malloc(blockDim.x * sizeof(bool)));
				buffA = static_cast<UInt*>(malloc(blockDim.x * sizeof(UInt)));
				buffB = static_cast<UInt*>(malloc(blockDim.x * sizeof(UInt)));

				warpBuffPos = static_cast<UInt*>(malloc(blockDim.x / 32 * sizeof(UInt)));
			}

			__syncthreads();

			if (warpThreadId == 0)
				warpBuffPos[warpThreadId] = 0;

			//uint start= wid        * ((inSize-1 ) / warpCount ) + max(0, - warpCount  + wid          + (inSize - 1) % warpCount ) + 1;
			int start = warpId * ((count - 1) / warpsCount) + max(0, -warpsCount + warpId + (count - 1) % warpsCount) + 1;
			//uint stop=(wid         + 1) * ((inSize-1 ) / warpCount ) + max(0, -warpCount  +  (inSize- 1) % warpCount  + wid          + 1);
			int stop = (warpId + 1) * ((count - 1) / warpsCount) + max(0, -warpsCount + (count - 1) % warpsCount + warpId + 1);

			if (warpId < warpsCount)
			{
				UInt outStart = outStarts[warpId];

				for (UInt i = start; i <= stop; i++)
				{
					float px = xCoords[i];
					float py = yCoords[i];

					flags[blockWarpId] = false;

					for (int j = i - 32; j >= -32; j -= 32)
					{
						int localId = warpThreadId + j;
						found[blockThreadId] = false;
						scanBuf[blockThreadId] = 0;

						if (localId >= 0)
						{
							if (instances[i].fields.featureId == instances[localId].fields.featureId)
								continue;

							float lx = xCoords[localId];

							if ((px - lx) > radius)
							{
								flags[blockWarpId] = true;
							}

							float ly = yCoords[localId];

							
							if ((MiningCommon::distance(px, py, lx, ly) <= radiusSquared))
							{
								found[blockThreadId] = true;

								if (instances[i].fields.featureId < instances[localId].fields.featureId
									|| instances[i].fields.instanceId < instances[localId].fields.instanceId)
								{
									temp_a = instances[i];
									temp_b = instances[localId];
								}
								else
								{
									temp_a = instances[localId];
									temp_b = instances[i];
								}

								scanBuf[blockThreadId] = 1;
							}
						}

						MiningCommon::intraWarpScan<UInt>(scanBuf);
						__syncthreads();

						//if (warpBuffPos[blockWarpId] + scanBuf)

						if (found[blockThreadId])
						{
							int pos = scanBuf[blockThreadId] + outStart;
							out_a[pos] = temp_a;
							out_b[pos] = temp_b;
						}

						outStart += scanBuf[blockWarpId * 32 + 31];

						/*
						scanBuf[blockThreadId] = found[blockThreadId];
						intraWarpScan(scanBuf);

						UInt oldLocalStart = localStart;

						if (found[blockThreadId])
						{
						UInt index = blockWarpId * 64 + (localStart + scanBuf[blockThreadId]) % 64;
						buffA[index] = temp_a.field;
						buffB[index] = temp_b.field;

						}

						// (
						//		localstart
						//		+ last value from scanbuff for last thread in warp
						//		+ last value from found for last thread in warp
						//	) mod 64
						localStart = (localStart + scanBuf[blockWarpId * 32 + 31] + found[blockWarpId * 32 + 31]) % 64;

						if (oldLocalStart < 32 && localStart >= 32)
						{
						out_a[outStart + warpThreadId].field = buffA[underBuffId];
						out_b[outStart + warpThreadId].field = buffB[underBuffId];
						outStart += 32;
						}
						else if (localStart < 32)
						{
						out_a[outStart + warpThreadId].field = buffA[aboveBuffId];
						out_b[outStart + warpThreadId].field = buffB[aboveBuffId];
						outStart += 32;
						}
						*/

						if (flags[blockWarpId])
						{
							break;
						}
					}
				}

				/*
				if (localStart < 32 && warpThreadId < localStart)
				{
				out_a[outStart + warpThreadId].field = buffA[underBuffId];
				out_b[outStart + warpThreadId].field = buffB[underBuffId];
				}
				else if (localStart >= 32 && warpThreadId < localStart - 32)
				{
				out_a[outStart + warpThreadId].field = buffA[aboveBuffId];
				out_b[outStart + warpThreadId].field = buffB[aboveBuffId];
				}
				*/
			}

			__syncthreads();

			if (threadIdx.x == 0)
			{
				free(const_cast<UInt*>(scanBuf));
				free(const_cast<bool*>(flags));
				free(const_cast<bool*>(found));
				//free(const_cast<UInt*>(buffA));
				//free(const_cast<UInt*>(buffB));
			}
		}
		// --------------------------------------------------------------------------------------------------------------------------------------

		__host__ void PlaneSweep(
			thrust::device_vector<float> xCoords
			, thrust::device_vector<float>& yCoords
			, thrust::device_vector<FeatureInstance>& instances
			, UInt count
			, float distanceTreshold
			, std::shared_ptr<GPUHashMapper<UInt, NeighboursListInfoHolder, GPUKeyProcessor<UInt>>>& resultHashMap
			, thrust::device_vector<FeatureInstance>& resultPairsA
			, thrust::device_vector<FeatureInstance>& resultPairsB)
		{


			UInt warpsCount = count;
			thrust::device_vector<UInt> neighboursCount(count);
			dim3 grid;

			findSmallest2D(warpsCount * 32, 256, grid.x, grid.y);

			countNeighbours <<< grid, 256 >>> (
				thrust::raw_pointer_cast(xCoords.data())
				, thrust::raw_pointer_cast(yCoords.data())
				, thrust::raw_pointer_cast(instances.data())
				, count
				, distanceTreshold
				, distanceTreshold * distanceTreshold
				, warpsCount
				, thrust::raw_pointer_cast(neighboursCount.data())
				);

			UInt totalPairsCount = neighboursCount[count - 1];
			thrust::exclusive_scan(neighboursCount.begin(), neighboursCount.end(), neighboursCount.begin());
			totalPairsCount += neighboursCount[count - 1];

			typedef thrust::device_vector<FeatureInstance> InstancesDeviceVector;

			resultPairsA = InstancesDeviceVector(totalPairsCount);
			resultPairsB = InstancesDeviceVector(totalPairsCount);

			findNeighbours <<< grid, 256 >>> (
				thrust::raw_pointer_cast(xCoords.data())
				, thrust::raw_pointer_cast(yCoords.data())
				, thrust::raw_pointer_cast(instances.data())
				, count
				, distanceTreshold
				, distanceTreshold * distanceTreshold
				, warpsCount
				, thrust::raw_pointer_cast(neighboursCount.data())
				, thrust::raw_pointer_cast(resultPairsA.data())
				, thrust::raw_pointer_cast(resultPairsB.data())
				);


			MiningCommon::zipSort(
				resultPairsA
				, resultPairsB
			);

			thrust::device_vector<FeatureInstance> uniques(totalPairsCount);
			thrust::device_vector<UInt> indices(totalPairsCount);
			thrust::device_vector<UInt> counts(totalPairsCount);

			UInt entryCount = thrust::reduce_by_key(
				resultPairsA.begin(),
				resultPairsA.end(),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						thrust::counting_iterator<UInt>(0),
						thrust::constant_iterator<UInt>(1)
					)
				),
				uniques.begin(),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						indices.begin(),
						counts.begin()
					)
				),
				MiningCommon::InstanceEquality<FeatureInstance>(),
				MiningCommon::FirstIndexAndCount<UInt>()
			).first - uniques.begin();

			constexpr float entryCountHashMapMultiplier = 1.5f;

			resultHashMap.reset(new GPUHashMapper<UInt, NeighboursListInfoHolder, GPUKeyProcessor<UInt>>(
				entryCount * entryCountHashMapMultiplier,
				new  GPUKeyProcessor<UInt>())
			);

			dim3 insertGrid;
			findSmallest2D(entryCount, 256, insertGrid.x, insertGrid.y);

			MiningCommon::InsertIntoHashMap << <insertGrid, 256 >> >(
				resultHashMap->getBean(),
				thrust::raw_pointer_cast(uniques.data())
				, thrust::raw_pointer_cast(indices.data())
				, thrust::raw_pointer_cast(counts.data()),
				entryCount
				);
		}
		// --------------------------------------------------------------------------------------------------------------------------------------

	}
}
