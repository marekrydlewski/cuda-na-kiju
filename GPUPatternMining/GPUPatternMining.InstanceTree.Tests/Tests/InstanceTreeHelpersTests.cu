#include "..\catch.hpp"
#include "..\BaseCudaTestHandler.h"

#define TEST_CUDA_CHECK_RETURN
//--------------------------------------------------------------

#include "..\..\GPUPatternMining/InstanceTree/InstanceTreeHelpers.h"
//--------------------------------------------------------------

using namespace InstanceTreeHelpers;
//--------------------------------------------------------------


TEST_CASE_METHOD(BaseCudaTestHandler, "Instance tree helpers | insert first pair count")
{
	thrust::device_vector<unsigned int> keys;
	{
		std::vector<unsigned int> hKeys
		{
			0x000A000B
			, 0x000A000C
			, 0x000B000C
		};

		keys = hKeys;
	}

	thrust::device_vector<Entities::InstanceTable> values;
	{
		std::vector<Entities::InstanceTable> hValues;

		Entities::InstanceTable it;

		it.count = 2;
		it.startIdx = 0;
		hValues.push_back(it);

		it.count = 3;
		it.startIdx = 2;
		hValues.push_back(it);

		it.count = 6;
		it.startIdx = 5;
		hValues.push_back(it);

		values = hValues;
	}
	
	auto proc = GPUUIntKeyProcessor();
	auto map = IntanceTablesMapCreator::InstanceTableMap(5, &proc);


	map.insertKeyValuePairs(
		thrust::raw_pointer_cast(keys.data())
		, thrust::raw_pointer_cast(values.data())
		, 3
	);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	thrust::device_vector<unsigned int> result(3);

	thrust::device_vector<thrust::device_vector<unsigned short>> cliquesData;
	thrust::host_vector<thrust::device_vector<unsigned short>> hcliquesData;
	{
		
		std::vector<unsigned short> first = { 0x000A, 0x000B, 0x000C };
		std::vector<unsigned short> second = { 0x000A, 0x000C, 0x000D };
		std::vector<unsigned short> third = { 0x000B, 0x000C, 0x000D };

		hcliquesData.push_back(first);
		hcliquesData.push_back(second);
		hcliquesData.push_back(third);

		cliquesData = hcliquesData;
	}

	thrust::device_vector<thrust::device_ptr<const unsigned short>> cliques;
	{
		std::vector<thrust::device_ptr<const unsigned short>> hcliques;
		for (const thrust::device_vector<unsigned short>& vec : hcliquesData)
			hcliques.push_back(vec.data());

		cliques = hcliques;
	}

	dim3 insertGrid;
	findSmallest2D(3, 256, insertGrid.x, insertGrid.y);
	
	fillFirstPairCountFromMap <<< insertGrid, 256 >>>(
		map.getBean()
		, thrust::raw_pointer_cast(cliques.data())
		, 3
		, result.data()
	);

	hipDeviceSynchronize();

	thrust::host_vector<unsigned int> hResult = result;

	std::vector<unsigned int> expected =
	{
		2, 3, 6
	};

	REQUIRE(std::equal(expected.begin(), expected.end(), hResult.begin()));
}
//--------------------------------------------------------------

TEST_CASE_METHOD(BaseCudaTestHandler, "Instance tree helpers | for groups simple")
{

	thrust::device_vector<unsigned int> counts;
	{
		std::vector<unsigned int> hCount = {
			2, 3, 2, 1
		};

		counts = hCount;
	}

	auto result = forGroups(counts);

	std::vector<unsigned int> expectedGroupNumbers = {
		0, 0, 1, 1, 1, 2, 2, 3
	};

	std::vector<unsigned int> expectedItemNumbers = {
		0, 1, 0, 1, 2, 0, 1, 0
	};

	thrust::host_vector<unsigned int> resultGroupNumbers = result->groupNumbers;
	thrust::host_vector<unsigned int> resultItemNumbers = result->itemNumbers;

	REQUIRE(std::equal(expectedGroupNumbers.begin(), expectedGroupNumbers.end(), resultGroupNumbers.begin()));
	REQUIRE(std::equal(expectedItemNumbers.begin(), expectedItemNumbers.end(), resultItemNumbers.begin()));
	REQUIRE(result->threadCount == 8);
}
//--------------------------------------------------------------
