#include "hip/hip_runtime.h"
#include "AnyLengthInstancesUniquePrevalenceProviderHelpers.h"
#include <thrust/execution_policy.h>


TypesCountsMapPtr getGpuTypesCountsMap(
	TypesCountsPtr typesCounts
	, GPUKeyProcessor<unsigned int>* mapKeyProcessor
)
{
	auto map = std::make_shared<TypesCountsMap>(typesCounts->size() * 1.5f, mapKeyProcessor);

	std::vector<unsigned int> values;
	std::vector<unsigned int> keys;

	for (TypeCount& tc : *typesCounts)
	{
		keys.push_back(tc.type);
		values.push_back(tc.count);
	}

	thrust::device_vector<unsigned int> gKeys = keys;
	thrust::device_vector<unsigned int> gValues = values;

	map->insertKeyValuePairs(
		gKeys.data().get()
		, gValues.data().get()
		, typesCounts->size()
	);

	return map;
}
// --------------------------------------------------------------------------------------------------

__global__
void fillTypesCountsForCliqueCandidatesInstances(
	TypesCountsMapBean bean
	, thrust::device_ptr<const unsigned short>* cliquesTypes
	, unsigned int count
	, unsigned int candidatesCount
	, thrust::device_ptr<unsigned int> typesCount
)
{
	unsigned int tid = computeLinearAddressFrom2D();
	
	if (tid < count)
	{
		unsigned int level = tid % candidatesCount;
		unsigned int pos = tid / candidatesCount;

		unsigned int key = cliquesTypes[level][pos];
		unsigned int val;

		GPUHashMapperProcedures::getValue(
			bean
			, key
			, val
		);

		typesCount[tid] = val;
	}
}
// --------------------------------------------------------------------------------------------------

UIntDeviceVectorPtr getTypesCountOnGpuForCliquesCandidates(
	Entities::GpuCliques cliquesCandidates
	, TypesCountsMapPtr typesCountsMap
)
{
	unsigned int threadCount = cliquesCandidates.candidatesCount * cliquesCandidates.currentCliquesSize;

	dim3 insertGrid;
	findSmallest2D(threadCount, 256, insertGrid.x, insertGrid.y);

	UIntDeviceVectorPtr result = std::make_shared<UIntDeviceVector>(threadCount);

	fillTypesCountsForCliqueCandidatesInstances << < insertGrid, 256 >> > (
		typesCountsMap->getBean()
		, cliquesCandidates.cliquesData->data().get()
		, threadCount
		, cliquesCandidates.candidatesCount
		, result->data()
		);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	return result;
}
// --------------------------------------------------------------------------------------------------

__host__ __device__
void MinimalCandidatePrevalenceCounter::operator()(unsigned int idx) const
{
	float currentMinimalPrevalence = 1;

	const unsigned int cliqueId = cliqueIds[idx];

	for (unsigned int currentLevel = 0; currentLevel < levelsCount; ++currentLevel)
	{
		float currentResult = thrust::distance
		(
			levelUniquesTempStorage + begins[idx]
			, thrust::unique_copy
			(
				thrust::device
				, data + (instancesCount * currentLevel) + begins[idx]
				, data + (instancesCount * currentLevel) + begins[idx] + counts[idx]
				, levelUniquesTempStorage + begins[idx]
			)
		) / static_cast<float>(typeCount[candidatesCount * currentLevel + cliqueId]);

		if (currentResult < currentMinimalPrevalence)
			currentMinimalPrevalence = currentResult;
	}

	results[cliqueId] = currentMinimalPrevalence;
}
// --------------------------------------------------------------------------------------------------
