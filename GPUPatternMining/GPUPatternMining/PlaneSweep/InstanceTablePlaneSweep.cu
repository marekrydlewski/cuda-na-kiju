#include "hip/hip_runtime.h"
#include "InstanceTablePlaneSweep.h"

#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "../Common/MiningCommon.h"

#include "../Entities/InstanceTable.h"
#include "../Common/CommonOperations.h"

#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>

namespace PlaneSweep
{
	namespace InstanceTable
	{
		__global__ void countNeighbours(
			float* xCoords
			, float* yCoords
			, FeatureInstance* instances
			, int count
			, float radius
			, float radiusSquared
			, int warpsCount
			, UInt* resultNeighboursCount)
		{
			// btid
			int blockThreadId = threadIdx.x;
			//gid
			int globalId = computeLinearAddressFrom2D();
			// wid
			int warpId = globalId / 32;
			// bwid
			int blockWarpId = blockThreadId / 32;
			// wtid
			int warpThreadId = threadIdx.x % 32;

			__shared__ volatile bool * flags;
			__shared__ volatile UInt * found;

			if (threadIdx.x == 0)
			{
				flags = static_cast<bool*>(malloc((blockDim.x / 32) * sizeof(bool)));
				found = static_cast<UInt*>(malloc(blockDim.x * uintSize));
			}

			__syncthreads();

			//uint start= wid        * ((inSize-1 ) / warpCount ) + max(0, - warpCount  + wid          + (inSize - 1) % warpCount ) + 1;
			int start = warpId * ((count - 1) / warpsCount) + max(0, -warpsCount + warpId + (count - 1) % warpsCount) + 1;
			//uint stop=(wid         + 1) * ((inSize-1 ) / warpCount ) + max(0, -warpCount  +  (inSize- 1) % warpCount  + wid          + 1);
			int stop = (warpId + 1) * ((count - 1) / warpsCount) + max(0, -warpsCount + warpId + 1 + (count - 1) % warpsCount);

			if (warpId == 63)
			{
				found[blockThreadId] = 0;
			}

			if (warpId < warpsCount)
			{
				found[blockThreadId] = 0;

				for (UInt i = start; i <= stop; i++)
				{
					float px = xCoords[i];
					float py = yCoords[i];

					flags[blockWarpId] = false;

					for (int j = i - 32; j >= -32; j -= 32)
					{
						int localId = warpThreadId + j;
						if (localId >= 0)
						{
							float lx = xCoords[localId];

							if ((px - lx) > radius)
							{
								flags[blockWarpId] = true;
							}

							float ly = yCoords[localId];

							if ((MiningCommon::distance(px, py, lx, ly) <= radiusSquared))
							{
								if (instances[i] != instances[localId])
									found[blockThreadId] += 1;
							}
						}

						if (flags[blockWarpId])
						{
							break;
						}
					}
				}

				MiningCommon::intraWarpReduce(found);

				// warpThreadId == 0?
				if (globalId == warpId * 32)
				{
					resultNeighboursCount[warpId] = found[blockWarpId * 32 + 31];
				}
			}

			__syncthreads();

			if (threadIdx.x == 0)
			{
				free(const_cast<bool*>(flags));
				free(const_cast<UInt*>(found));
			}
		}
		// --------------------------------------------------------------------------------------------------------------------------------------

		__global__ void findNeighbours(
			float* xCoords
			, float* yCoords
			, FeatureInstance* instances
			, int count
			, float radius
			, float radiusSquared
			, int warpsCount
			, UInt *outStarts
			, FeatureInstance* out_a
			, FeatureInstance* out_b)
		{
			// btid
			int blockThreadId = threadIdx.x;
			//gid
			int globalId = computeLinearAddressFrom2D();
			// wid
			int warpId = globalId / 32;
			// bwid
			int blockWarpId = blockThreadId / 32;
			// wtid
			int warpThreadId = threadIdx.x % 32;

			// const UInt underBuffId = blockWarpId * 64 + warpThreadId;
			// const int aboveBuffId = blockWarpId * 64 + 32 + warpThreadId;

			__shared__ volatile UInt* scanBuf;
			__shared__ volatile bool* flags;
			__shared__ volatile bool* found;
			//volatile __shared__ UInt* buffA;
			//volatile __shared__ UInt* buffB;
			__shared__ UInt* warpBuffPos;


			FeatureInstance temp_a;
			FeatureInstance temp_b;
			// UInt localStart = 0;

			if (threadIdx.x == 0)
			{
				// check Allocating http://www.drdobbs.com/parallel/a-massively-parallel-stack-for-data-allo/240162018?pgno=1
				// measure dynamic allocating in different warps
				scanBuf = static_cast<UInt*>(malloc(blockDim.x * uintSize));
				flags = static_cast<bool*>(malloc((blockDim.x / 32) * sizeof(bool)));
				found = static_cast<bool*>(malloc(blockDim.x * sizeof(bool)));
				//buffA = static_cast<UInt*>(malloc(blockDim.x * sizeof(UInt)));
				//buffB = static_cast<UInt*>(malloc(blockDim.x * sizeof(UInt)));

				warpBuffPos = static_cast<UInt*>(malloc(blockDim.x / 32 * sizeof(UInt)));
			}

			__syncthreads();

			if (warpThreadId == 0)
				warpBuffPos[warpThreadId] = 0;

			//uint start= wid        * ((inSize-1 ) / warpCount ) + max(0, - warpCount  + wid          + (inSize - 1) % warpCount ) + 1;
			int start = warpId * ((count - 1) / warpsCount) + max(0, -warpsCount + warpId + (count - 1) % warpsCount) + 1;
			//uint stop=(wid         + 1) * ((inSize-1 ) / warpCount ) + max(0, -warpCount  +  (inSize- 1) % warpCount  + wid          + 1);
			int stop = (warpId + 1) * ((count - 1) / warpsCount) + max(0, -warpsCount + (count - 1) % warpsCount + warpId + 1);

			if (warpId < warpsCount)
			{
				UInt outStart = outStarts[warpId];

				for (UInt i = start; i <= stop; i++)
				{
					float px = xCoords[i];
					float py = yCoords[i];

					flags[blockWarpId] = false;

					for (int j = i - 32; j >= -32; j -= 32)
					{
						int localId = warpThreadId + j;
						found[blockThreadId] = false;
						scanBuf[blockThreadId] = 0;

						if (localId >= 0)
						{
							float lx = xCoords[localId];

							if ((px - lx) > radius)
							{
								flags[blockWarpId] = true;
							}

							float ly = yCoords[localId];

							if (instances[i] != instances[localId])
							{
								if ((MiningCommon::distance(px, py, lx, ly) <= radiusSquared))
								{
									found[blockThreadId] = true;

									if (instances[i].fields.featureId < instances[localId].fields.featureId
										|| instances[i].fields.instanceId < instances[localId].fields.instanceId)
									{
										temp_a = instances[i];
										temp_b = instances[localId];
									}
									else
									{
										temp_a = instances[localId];
										temp_b = instances[i];
									}

									scanBuf[blockThreadId] = 1;
								}
							}
						}

						MiningCommon::intraWarpScan<UInt>(scanBuf);
						__syncthreads();

						//if (warpBuffPos[blockWarpId] + scanBuf)

						if (found[blockThreadId])
						{
							int pos = scanBuf[blockThreadId] + outStart;
							out_a[pos] = temp_a;
							out_b[pos] = temp_b;
						}

						outStart += scanBuf[blockWarpId * 32 + 31];

						/*
						scanBuf[blockThreadId] = found[blockThreadId];
						intraWarpScan(scanBuf);

						UInt oldLocalStart = localStart;

						if (found[blockThreadId])
						{
						UInt index = blockWarpId * 64 + (localStart + scanBuf[blockThreadId]) % 64;
						buffA[index] = temp_a.field;
						buffB[index] = temp_b.field;

						}

						// (
						//		localstart
						//		+ last value from scanbuff for last thread in warp
						//		+ last value from found for last thread in warp
						//	) mod 64
						localStart = (localStart + scanBuf[blockWarpId * 32 + 31] + found[blockWarpId * 32 + 31]) % 64;

						if (oldLocalStart < 32 && localStart >= 32)
						{
						out_a[outStart + warpThreadId].field = buffA[underBuffId];
						out_b[outStart + warpThreadId].field = buffB[underBuffId];
						outStart += 32;
						}
						else if (localStart < 32)
						{
						out_a[outStart + warpThreadId].field = buffA[aboveBuffId];
						out_b[outStart + warpThreadId].field = buffB[aboveBuffId];
						outStart += 32;
						}
						*/

						if (flags[blockWarpId])
						{
							break;
						}
					}
				}

				/*
				if (localStart < 32 && warpThreadId < localStart)
				{
				out_a[outStart + warpThreadId].field = buffA[underBuffId];
				out_b[outStart + warpThreadId].field = buffB[underBuffId];
				}
				else if (localStart >= 32 && warpThreadId < localStart - 32)
				{
				out_a[outStart + warpThreadId].field = buffA[aboveBuffId];
				out_b[outStart + warpThreadId].field = buffB[aboveBuffId];
				}
				*/
			}

			__syncthreads();

			if (threadIdx.x == 0)
			{
				free(const_cast<UInt*>(scanBuf));
				free(const_cast<bool*>(flags));
				free(const_cast<bool*>(found));
				//free(const_cast<UInt*>(buffA));
				//free(const_cast<UInt*>(buffB));
			}
		}
		// --------------------------------------------------------------------------------------------------------------------------------------

		typedef thrust::tuple<FeatureInstance, FeatureInstance> FeatureInstanceTuple;

		typedef thrust::device_vector<FeatureInstance>::iterator FeatureDeviceVectorIterator;
		typedef thrust::tuple<FeatureDeviceVectorIterator, FeatureDeviceVectorIterator> FeatureInstanceIteratorTuple;
		typedef thrust::zip_iterator<FeatureInstanceIteratorTuple> FeatureInstanceTupleIterator;
		// --------------------------------------------------------------------------------------------------------------------------------------

		struct FeatureInstanceTupleEquality : public thrust::binary_function<FeatureInstanceTuple, FeatureInstanceTuple, bool>
		{
			__host__ __device__ bool operator()(const FeatureInstanceTuple& lhs, const FeatureInstanceTuple& rhs) const
			{
				return lhs.get<0>().fields.featureId == rhs.get<0>().fields.featureId
					&& lhs.get<1>().fields.featureId == rhs.get<1>().fields.featureId;
			}
		};
		//---------------------------------------------------------------------------------------------

		__global__ void InsertFeatureInstanceTupleIntoHashMap(
			HashMapperBean<unsigned int, Entities::InstanceTable, GPUUIntKeyProcessor> bean,
			FeatureInstanceTuple* keys,
			unsigned int* deltas,
			unsigned int* counts,
			unsigned int count
		)
		{
			unsigned int tid = computeLinearAddressFrom2D();

			if (tid < count)
			{
				GPUHashMapperProcedures::insertKeyValuePair(
					bean,
					(keys[tid].get<0>().field & 0xFFFF0000) | (keys[tid].get<1>().field >> 16) ,
					Entities::InstanceTable(counts[tid], deltas[tid])
				);
			}
		}
		//---------------------------------------------------------------------------------------------

		__host__ void PlaneSweep(
			thrust::device_vector<float> xCoords
			, thrust::device_vector<float>& yCoords
			, thrust::device_vector<FeatureInstance>& instances
			, UInt count
			, float distanceTreshold
			, PlaneSweepTableInstanceResultPtr result)
		{
			UInt warpsCount = count;
			thrust::device_vector<UInt> neighboursCount(count);
			dim3 grid;

			findSmallest2D(warpsCount * 32, 256, grid.x, grid.y);

			countNeighbours <<< grid, 256 >>> (
				thrust::raw_pointer_cast(xCoords.data())
				, thrust::raw_pointer_cast(yCoords.data())
				, thrust::raw_pointer_cast(instances.data())
				, count
				, distanceTreshold
				, distanceTreshold * distanceTreshold
				, warpsCount
				, thrust::raw_pointer_cast(neighboursCount.data())
				);

			CUDA_CHECK_RETURN(hipDeviceSynchronize());
			
			UInt totalPairsCount = neighboursCount[count - 1];
			thrust::exclusive_scan(neighboursCount.begin(), neighboursCount.end(), neighboursCount.begin());
			totalPairsCount += neighboursCount[count - 1];

			typedef thrust::device_vector<FeatureInstance> InstancesDeviceVector;

			result->pairsA = InstancesDeviceVector(totalPairsCount);
			result->pairsB = InstancesDeviceVector(totalPairsCount);

			findNeighbours <<< grid, 256 >>> (
				thrust::raw_pointer_cast(xCoords.data())
				, thrust::raw_pointer_cast(yCoords.data())
				, thrust::raw_pointer_cast(instances.data())
				, count
				, distanceTreshold
				, distanceTreshold * distanceTreshold
				, warpsCount
				, thrust::raw_pointer_cast(neighboursCount.data())
				, thrust::raw_pointer_cast(result->pairsA.data())
				, thrust::raw_pointer_cast(result->pairsB.data())
				);

			CUDA_CHECK_RETURN(hipDeviceSynchronize());

			MiningCommon::zipSort(
				result->pairsA
				, result->pairsB
			);			

			FeatureInstanceTupleIterator zippedBegin = thrust::make_zip_iterator(thrust::make_tuple(
				result->pairsA.begin()
				, result->pairsB.begin()
			));
			
			FeatureInstanceTupleIterator zippedEnd = thrust::make_zip_iterator(thrust::make_tuple(
				result->pairsA.end()
				, result->pairsB.end()
			));
			
			result->uniques = thrust::device_vector<FeatureInstanceTuple>(totalPairsCount);
			result->indices = thrust::device_vector<UInt>(totalPairsCount);
			result->counts = thrust::device_vector<UInt>(totalPairsCount);
			
			UInt entryCount = thrust::reduce_by_key(
				zippedBegin,
				zippedEnd,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						thrust::counting_iterator<UInt>(0),
						thrust::constant_iterator<UInt>(1)
					)
				),
				result->uniques.begin(),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						result->indices.begin(),
						result->counts.begin()
					)
				),
				FeatureInstanceTupleEquality(),
				MiningCommon::FirstIndexAndCount<UInt>()
			).first - result->uniques.begin();
			
			constexpr float entryCountHashMapMultiplier = 1.5f;

			result->instanceTableMap.reset(new GPUHashMapper<UInt, Entities::InstanceTable, GPUKeyProcessor<UInt>>(
				entryCount * entryCountHashMapMultiplier,
				new  GPUKeyProcessor<UInt>())
			);

			dim3 insertGrid;
			findSmallest2D(entryCount, 256, insertGrid.x, insertGrid.y);
			
			InsertFeatureInstanceTupleIntoHashMap <<< insertGrid, 256 >>>(
				result->instanceTableMap->getBean(),
				thrust::raw_pointer_cast(result->uniques.data())
				, thrust::raw_pointer_cast(result->indices.data())
				, thrust::raw_pointer_cast(result->counts.data()),
				entryCount
				);

			CUDA_CHECK_RETURN(hipDeviceSynchronize());
		}
		// --------------------------------------------------------------------------------------------------------------------------------------

	}
}
