#include "GpuMiningAlgorithm.h"

#include <algorithm>

#include "../GPUPatternMining/PlaneSweep/PlaneSweepTableInstanceResult.h"
#include "../GPUPatternMining/PlaneSweep/InstanceTablePlaneSweep.h"
#include "../GPUPatternMining/Prevalence/PrevalentTypedPairProvider.h"
#include "../GPUPatternMining/Prevalence/AnyLengthInstancesUniquePrevalenceProvider.h"

// ------------------------------------------------------------------------------------------------


void GpuMiningAlgorithm::loadData(DataFeed * data, size_t size, unsigned short types)
{

	typeIncidenceCounter = std::make_shared<TypesCounts>(types, TypeCount(0,0));
	source.assign(data, data + size);
}

void GpuMiningAlgorithm::filterByDistancePrepareData()
{
	filterByDistanceGpuData = std::make_shared<FilterByDistanceGpuData>();

	std::vector<float> hx(source.size());
	std::transform(source.begin(), source.end(), hx.begin(),
		[](DataFeed& df)
	{
		return df.xy.x;
	});

	filterByDistanceGpuData->x = hx;

	std::vector<float> hy(source.size());
	std::transform(source.begin(), source.end(), hy.begin(),
		[](DataFeed& df)
	{
		return df.xy.y;
	});

	filterByDistanceGpuData->y = hy;;

	std::vector<FeatureInstance> hInstances(source.size());
	std::transform(source.begin(), source.end(), hInstances.begin(),
		[](DataFeed& df)
	{
		FeatureInstance fi;
		fi.fields.featureId = df.type;
		fi.fields.instanceId = df.instanceId;
		return fi;
	});

	filterByDistanceGpuData->instances = hInstances;

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	PlaneSweep::InstanceTable::SortByXAxis(
		filterByDistanceGpuData->x
		, filterByDistanceGpuData->y
		, filterByDistanceGpuData->instances
	);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

}

void GpuMiningAlgorithm::filterByDistance(float threshold)
{
	planeSweepResult = std::make_shared<PlaneSweepTableInstanceResult>();

	PlaneSweep::InstanceTable::PlaneSweep(
		filterByDistanceGpuData->x
		, filterByDistanceGpuData->y
		, filterByDistanceGpuData->instances
		, filterByDistanceGpuData->instances.size()
		, threshold
		, planeSweepResult
	);

	filterByDistanceGpuData.reset();
}

void GpuMiningAlgorithm::filterPrevalentTypedConnectionsPrepareData()
{
	itmPack = IntanceTablesMapCreator::createTypedNeighboursListMap(
		planeSweepResult->pairsA
		, planeSweepResult->pairsB
	);

	// TODO write kernel

	for (unsigned int i = 0; i < typeIncidenceCounter->size(); ++i)
		typeIncidenceCounter->at(i).type = i;

	
	for (DataFeed& df : source)
		typeIncidenceCounter->at(df.type).count += 1;
}

void GpuMiningAlgorithm::filterPrevalentTypedConnections(float minimalPrevalence)
{
	Prevalence::UniqueFilter::PrevalentTypedPairProvider bppc(
		*typeIncidenceCounter, itmPack);

	prevalentTypesConnections = bppc.getPrevalentPairConnections(
		minimalPrevalence
		, planeSweepResult
	);
}

void GpuMiningAlgorithm::constructMaximalCliquesPrepareData()
{
	graphForKerbosh.setSize(typeIncidenceCounter->size());
	unsigned int edgeCount = 0;

	for (FeatureTypePair& ftp : prevalentTypesConnections)
	{
		graphForKerbosh.addEdge(ftp.types.a, ftp.types.b);
		++edgeCount;
	}

	printf("Typed graph now contains %u edges\n", edgeCount);
}

void GpuMiningAlgorithm::constructMaximalCliques()
{
	CliquesContainer pendingCliques;

	auto degeneracy = graphForKerbosh.getDegeneracy();
	for (unsigned short const vertex : degeneracy.second)
	{
		std::vector<unsigned short> neighboursWithHigherIndices = graphForKerbosh.getVertexNeighboursOfHigherIndex(vertex);
		std::vector<unsigned short> neighboursWithLowerIndices = graphForKerbosh.getVertexNeighboursOfLowerIndex(vertex);
		std::vector<unsigned short> thisVertex = { vertex };

		auto generatedCliques = graphForKerbosh.bkPivot(
			neighboursWithHigherIndices,
			thisVertex,
			neighboursWithLowerIndices);

		for (std::vector<unsigned short >& cnd : generatedCliques)
		{
			if (pendingCliques.checkCliqueExistence(cnd))
				continue;

			candidates[cnd.size()].push_back(cnd);
			pendingCliques.insertClique(cnd);
		}
	}
}

void GpuMiningAlgorithm::filterCandidatesByPrevalencePrepareData()
{
	ITNMPack = InstanceTypedNeighboursMapCreator::createTypedNeighboursListMap(
		planeSweepResult->pairsA
		, planeSweepResult->pairsB
	);

	keyProc = std::make_shared<GPUUIntKeyProcessor>();

	typesCountsMap = getGpuTypesCountsMap(typeIncidenceCounter, keyProc.get());

	anyLengthPrevalenceProvider = std::make_shared<AnyLengthInstancesUniquePrevalenceProvider>(typesCountsMap);
	
	instanceTree = std::make_shared<InstanceTree::InstanceTree>(
		planeSweepResult
		, itmPack
		, ITNMPack
	);
}


std::vector<std::vector<unsigned short>> getAllCliquesSmallerByOne(std::vector<unsigned short>& clique)
{
	std::vector<std::vector<unsigned short>> smallCliques;
	for (auto i = 0; i < clique.size(); ++i)
	{
		std::vector<unsigned short> smallClique;
		for (auto j = 0; j < clique.size(); ++j)
		{
			if (j != i) smallClique.push_back(clique[j]);
		}
		smallCliques.push_back(smallClique);
	}
	return smallCliques;
}

std::list<std::vector<unsigned short>> GpuMiningAlgorithm::filterCandidatesByPrevalence(float minimalPrevalence)
{
	std::list<std::vector<unsigned short>> result;

	for (auto cands = candidates.rbegin(); cands  != candidates.rend(); ++cands)
	{
		std::vector<std::vector<unsigned short>> toProcess;
		
		{
			// for removing candidates repeating with earlier candidates
			CliquesContainer pendingCliques;

			for (auto cand : cands->second)
			{
				if (prevalentCliques.checkCliqueExistence(cand) || pendingCliques.checkCliqueExistence(cand))
					continue;

				pendingCliques.insertClique(cand);
				toProcess.push_back(cand);
			}
		}

		if (toProcess.empty())
			continue;

		unsigned int currentCliqueSize = toProcess[0].size();

		if (currentCliqueSize < 2)
			continue;

		auto gpuCliques = Entities::moveCliquesCandidatesToGpu(toProcess);

		auto instanceTreeResult = instanceTree->getInstancesResult(gpuCliques);

		auto mask = anyLengthPrevalenceProvider->getPrevalenceFromCandidatesInstances(
			gpuCliques
			, instanceTreeResult
		);

		thrust::host_vector<float> hPrevalences = *mask;
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		
		{
			// for removing new repeating candidates
			CliquesContainer pendingCliques;

			for (int i = 0; i < hPrevalences.size(); ++i)
			{
				if (hPrevalences[i] >= minimalPrevalence)
				{
					prevalentCliques.insertClique(toProcess[i]);

					result.push_back(toProcess[i]);
				}
				else if (currentCliqueSize > 2)
				{
					auto smallerCliques = getAllCliquesSmallerByOne(toProcess[i]);

					for (auto cand : smallerCliques)
					{
						if (pendingCliques.checkCliqueExistence(cand) || prevalentCliques.checkCliqueExistence(cand))
							continue;

						candidates[currentCliqueSize - 1].push_back(cand);
						pendingCliques.insertClique(cand);
					}
				}
			}
		}
	}

	return result;
}